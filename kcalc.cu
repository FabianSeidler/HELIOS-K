#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <sys/time.h>
#include <sys/resource.h>

#include "define.h"
#include "ISO.h"

// *********************************************
//This function calculates the Series Sigma1. Sigma2 and Sigma3 (Equations 27, 28, and 29) from Alg 916
//The parameter TOL sets a tolerance where to truncate the series
//It returns the values for sigma1 sigma2 and sigma3
//
// This implementation has still problems with loss of accuracy
//
//
//Author Simon Grimm, Adapted from Zaghloul & Ali, Algorithm 916
//November 2014
// **********************************************
__device__ void Sigma(double x, double y, double &s1, double &s2, double &s3, double a, double ex2, int id){

	s1 = 0.0;
	double sold1 = s1;
	s2 = 0.0;
	double sold2 = s2;
	s3 = 0.0;
	double sold3 = s3;

	double f, f3p, f3n;
	double an, an3p, an3n;

	double yy = y * y;

	int n0 = (int)(ceil(x / a)); //starting point for sigma3 series
	int n3p, n3n;

	int stop1 = 0;
	int stop2 = 0;
	int stop3 = 0;

	double ean2;

	double e2axn = exp(-2.0 * a * x);
	double e2axns = 1.0;

	double an0 = a * n0;
	double e3p = exp(-2.0 * a * (an0 - x - a));
	double e3n = exp(2.0 * a * (an0 - x));

	double e3ps = exp(-(an0 * an0 - 2.0 * a * an0 - 2.0 * an0 * x + x * x + a * a + 2.0 * a * x));
	double e3ns = exp(-(an0 * an0 - 2.0 * an0 * x + x * x));

	double st;

	for(int n = 1; n < 100; ++n){
		n3p = n0 + n - 1;
		n3n = n0 - n;
		an = a * n;
		an3p = a * n3p;
		an3n = a * n3n;

		ean2 = exp(-an * an);
		e2axns *= e2axn;	

		e3ps *= e3p;
		e3ns *= e3n;

		f = 1.0 / (an * an + yy);
		f3p = 1.0 / (an3p * an3p + yy);
		f3n = 1.0 / (an3n * an3n + yy);

		st = f * ean2 * ex2;

		s1 += st;
		s2 += st * e2axns;
		s3 += f3p * ean2 * e3ps;
		if(n3n >= 1) s3 += f3n * ean2 * e3ns;

		
		if(fabs(s1 - sold1) < TOL) stop1 = 1;		
		if(fabs(s2 - sold2) < TOL) stop2 = 1;		
		if(fabs(s3 - sold3) < TOL) stop3 = 1;		
		if(stop1 == 1 && stop2 ==1 && stop3 == 1) break;

		sold1 = s1;
		sold2 = s2;
		sold3 = s3;
//if(n >= 100-1) printf("Sigma Series did not converge\n");
	}
}
// *********************************************
//This function calculates the Series Sigma1. Sigma2 and Sigma3 (Equations 15, 16, and 17) from Alg 916
//The parameter TOL sets a tolerance where to truncate the series
//It returns the values for sigma1 sigma2 and sigma3

//Author Simon Grimm, Adapted from Zaghloul & Ali, Algorithm 916
//November 2014
// **********************************************
__device__ void Sigmab(double x, double y, double &s1, double &s2, double &s3, double a, double ex2, int id){

	s1 = 0.0;
	double sold1 = s1;
	s2 = 0.0;
	double sold2 = s2;
	s3 = 0.0;
	double sold3 = s3;

	double f, f3p, f3n;
	double an, an3p, an3n;

	double yy = y * y;

	if(x < 0.0) x = -x;

		int n0 = (int)(ceil(x / a)); //starting point for sigma3 series
		int n3p, n3n;

		int stop1 = 0;
		int stop2 = 0;
		int stop3 = 0;

		double e2axn = exp(-2.0 * a * x);

		for(int n = 1; n < 100; ++n){
		n3p = n0 + n - 1;
		n3n = n0 - n;
		an = a * n;
		an3p = a * n3p;
		an3n = a * n3n;

		f = 1.0 / (an * an + yy);
		f3p = 1.0 / (an3p * an3p + yy);
		f3n = 1.0 / (an3n * an3n + yy);

		s1 += f * exp(-(an * an + x * x));
		s2 += f * exp(-(an + x) * (an + x));
		s3 += f3p * exp(-(an3p - x) * (an3p - x));
		if(n3n >= 1) s3 += f3n * exp(-(an3n - x) * (an3n - x));

		if(fabs(s1 - sold1) < TOL) stop1 = 1;
		if(fabs(s2 - sold2) < TOL) stop2 = 1;
		if(fabs(s3 - sold3) < TOL) stop3 = 1;
		if(stop1 == 1 && stop2 ==1 && stop3 == 1) break;

		sold1 = s1;
		sold2 = s2;
		sold3 = s3;
//		if(n >= 100-1) printf("Sigma Series did not converge %d\n", id);
	}
}
// **************************************************
//This function calculates the Voigt profile V(x,y) as equation 13 from Zaghloul & Ali, Algorithm 916
//it calls the Sigma function
//The parameter TOL sets a tolerance where to truncate the series

//Author Simon Grimm, Adapted from Zaghloul & Ali, Algorithm 916
//November 2014
// **********************************************
__device__ double voigt_916(double x, double y, double a, int id){

	double s1, s2, s3;
	double ex2 = exp(-x * x);

	//Compute Sigma Series
	if(x != 0.0 && y != 0.0) Sigmab(x, y, s1, s2, s3, a, ex2, id);

	double xy = x * y;
	double a2ipi = 2.0 * a / M_PI;
	double cos2xy = cos(2.0 * xy);
	double sinxy = sin(xy);

	double t1;
	t1 = ex2 * erfcx(y) * cos2xy;
	t1 += a2ipi * x * sinxy * ex2 * sinxy / xy;
	t1 += a2ipi * y * (-cos2xy * s1 + 0.5 * (s2 + s3));
	
	if(x == 0) t1 = erfcx(y);
	if(y == 0) t1 = exp(-x * x);
	//if(x*x + y*y > 1.0e18) t1 = y / (sqrt(M_PI) * (x * x + y * y));
	
	return t1;

}
// **************************************************
//This kernel calculates the integrate line strength, the Lorentz and the Doppler halfwidths
//
//Author Simon Grimm
//November 2014
// **********************************************
__global__ void S_kernel(double *nu_d, double *S_d, double *A_d, double *EL_d, double *alphaL_d, double *alphaD_d, double *n_d, double *mass_d, double *gamma_d, double *delta_d, double *Q_d, int NL, double T, double P, int kk){

	int idx = threadIdx.x;
	int id = blockIdx.x * blockDim.x + idx + kk;

	if(id < NL){
		double c2 = c * c;
		double m = mass_d[id] / NA;			// mass in g

		double nu = nu_d[id] + delta_d[id] * P;		//read nu from alphaD
		nu_d[id] = nu;
		double S = S_d[id] / m;				//cm / g
		double EL = EL_d[id];  				//1/cm
		double Q = Q_d[id];				//Q0 / Q(T)
		double alphaL = alphaL_d[id];
		
		S_d[id] = S * Q * exp(-EL * h * c / (kB * T) + EL * h * c / (kB * T0)) * (1.0 - exp(-h * nu * c / (kB * T))) / (1.0 - exp(-h * nu * c / (kB * T0))); 
		alphaD_d[id] = 1.0 / (nu * sqrt(2.0 * kB * T / (m * c2)));	//inverse Doppler halfwith
		alphaL *= P * pow(T0 / T, gamma_d[id]);
		alphaL += A_d[id] / (4.0 * M_PI * c);				//1/cm
		alphaL_d[id] = alphaL;
	}
}

// **************************************************
//This kernel directly calls the Voigt function
//
//Author Simon Grimm
//November 2014
// **********************************************
__global__ void Voigt_line_kernel(double a, double dnu, double *K_d, double Nx, double xmax){

	int idx = threadIdx.x;
	int id = blockIdx.x * blockDim.x + idx;

	if(id < Nx){
		double aTOL = M_PI * sqrt(-1.0 / log(TOL * 0.5));
		double x = fabs(-xmax + id * 2.0 * xmax / ((double)(Nx)));
		K_d[id] = voigt_916(x, a, aTOL, id);
	}
}

// **************************************************
//This kernel computes the line shape
// It uses patterns of shared memory the reduce global memory access
//
//Author Simon Grimm
//November 2014
// **********************************************
template <int NB, int nl>
__global__ void Line_kernel(double *nu_d, double *S_d, double *alphaL_d, double *alphaD_d, double *n_d, double *K_d, double dnu, double numin, int Nx, int NL, int ii, int kk){

	int idx = threadIdx.x;
	int id = blockIdx.x * blockDim.x + idx + kk;

	__shared__ double nu_s[NB];
	__shared__ double S_s[NB];
	__shared__ double alphaL_s[NB];
	__shared__ double ialphaD_s[NB];
	__shared__ int xyFlag_s[2];

	double K = 0.0;

	double a = M_PI * sqrt(-1.0 / log(TOL * 0.5));
	double sqln2 = sqrt(log(2.0));

	double isqrtpi = 1.0 / sqrt(M_PI);
	double nu = numin + id * dnu;

	for(int i = 0; i < nl; i += NB){
		xyFlag_s[0] = 0;
		xyFlag_s[1] = 0;
		if(i + idx + ii < NL){
			nu_s[idx] = nu_d[i + idx + ii];
			S_s[idx] = S_d[i + idx + ii];
			alphaL_s[idx] = alphaL_d[i + idx + ii];
			ialphaD_s[idx] = alphaD_d[i + idx + ii];
		}
		else{
			nu_s[idx] = 0.0;
			S_s[idx] = 0.0;
			alphaL_s[idx] = 0.0;
			ialphaD_s[idx] = 0.0;
		}
		__syncthreads();

# if PROFILE == 1
		//Check smallest values for x and y
		for(int j = 0; j < NB; ++j){
			if(i + j < NL){
				double x = sqln2 * fabs((nu - nu_s[j]) * ialphaD_s[j]);
				double y = sqln2 * alphaL_s[j] * ialphaD_s[j];
				if(x * x + y * y < 1e6) xyFlag_s[1] = 1; 
				if(x * x + y * y < 100) xyFlag_s[0] = 1; 
			}
		}
		__syncthreads();
		if(xyFlag_s[0] == 1){
			for(int j = 0; j < NB; ++j){
				if(i + j < NL){
					double x = sqln2 * fabs((nu - nu_s[j]) * ialphaD_s[j]);
					double y = sqln2 * alphaL_s[j] * ialphaD_s[j];
					K += S_s[j] * voigt_916(x, y, a, id) * sqln2 * ialphaD_s[j] * isqrtpi;
//if(id < 32) printf("%d %.20g %.20g %.20g %.20g %.20g %.20g\n", id, nu, x, y, nu_s[j], 1.0 / ialphaD_s[j], alphaL_s[j]);
				}

			}
		}
		else if(xyFlag_s[1] == 1){
			for(int j = 0; j < NB; ++j){
				if(i + j < NL){
					//2nd order Gauss Hermite Quadrature
					double x = sqln2 * fabs((nu - nu_s[j]) * ialphaD_s[j]);
					double y = sqln2 *  alphaL_s[j] * ialphaD_s[j];
					double xxyy = x * x + y * y;
					double t = y / 3.0;
					double t1 = 2.0 * t / (M_PI * xxyy);
					double t2 = t * (xxyy + 1.5) / (M_PI * (xxyy + 1.5) * (xxyy + 1.5) - 4.0 * x * x * 1.5);
					K += S_s[j] * sqln2 * ialphaD_s[j] * (t1 + t2);
				}
			}
		}
		else{
			for(int j = 0; j < NB; ++j){
				if(i + j < NL){
					//1 order Gauss Hermite Quadrature
					double x = sqln2 * fabs((nu - nu_s[j]) * ialphaD_s[j]);
					double y = sqln2 *  alphaL_s[j] * ialphaD_s[j];
					K += S_s[j] * sqln2 * y * ialphaD_s[j] / (M_PI * (x * x + y * y));
				}
			}
		}
#endif
# if PROFILE == 2
		for(int j = 0; j < NB; ++j){
			if(i + j < NL){
				K += S_s[j] * alphaL_s[j] / (M_PI * ((nu - nu_s[j]) * (nu - nu_s[j]) + alphaL_s[j] * alphaL_s[j]));
			}	
		}
#endif
# if PROFILE == 3
		for(int j = 0; j < NB; ++j){
			if(i + j < NL){
				K += S_s[j] * ialphaD_s[j] * isqrtpi * exp(-x * x);
			}	
		}
#endif
		__syncthreads();
	}
	if(id < Nx) K_d[id] += K;
}

// ****************************************
// This function computes the Chebyshev polynomials as a function of T
// n must by greater than 1.
// *****************************************
__host__ void Chebyshev(double T, double *Cheb, int n){
	Cheb[0] = 1.0;
	Cheb[1] = T;
	for(int i = 2; i < n; ++i){
		Cheb[i] = 2.0 * T * Cheb[i - 1] - Cheb[i - 2];
	}
	
}
// ****************************************
// This function reads the file q.dat and computes for each Isotopologue
// the corresponding Partition function Q(T)

//Author: Simon Grimm
//November 2014
// *****************************************
__host__ int ChebCoeff(char *qFilename, Partition &part, double T){

	//Calculate Chebychev polynomial
	double Cheb[NCheb];
	Chebyshev(T, Cheb, NCheb);

	//Read Chebychev Coefficients from q file	
	FILE *qFile;
	//Check size of q.dat file
	qFile = fopen(qFilename, "r");
	if(qFile == NULL){
		printf("Error: q.dat file not found\n");
		return 0;
	}
	int j;
	for(j = 0; j < 100; ++j){
		int id;
		double coeff;
		int er = fscanf (qFile, "%d", &id);
		if (er <= 0) break;
		for(int i = 0; i < NCheb; ++i){
			fscanf (qFile, "%lf", &coeff);
		}
	}
	fclose(qFile);
	part.n = j;
	
	part.id = (int*)malloc(j * sizeof(int));
	part.Q = (double*)malloc(j * sizeof(double));
	
	qFile = fopen(qFilename, "r");
	for(j = 0; j < 100; ++j){
		int id;
		double coeff;
		double Q = 0.0;
		int er = fscanf (qFile, "%d", &id);
		if (er <= 0) break;
		for(int i = 0; i < NCheb; ++i){
			fscanf (qFile, "%lf", &coeff);
			Q += coeff * Cheb[i];
		}
		part.id[j] = id;
		part.Q[j] = Q;
	}
	fclose(qFile);
	return 1;
}

__host__ int read_parameters(Param &param, char *paramFilename, int argc, char*argv[]){
	//Read parameters from param.dat file
	FILE *paramFile;
	param.dev = 0;
	paramFile = fopen(paramFilename, "r");
		char skip[160];
		char skip2[160];
		//read T
		fgets(skip, 4, paramFile);
		fscanf (paramFile, "%lf", &param.T);
		fgets(skip2, 3, paramFile);
		//read P
		fgets(skip, 4, paramFile);
		fscanf (paramFile, "%lf", &param.P);
		fgets(skip2, 3, paramFile);
		//read Molecule
		fgets(skip, 11, paramFile);
		fscanf (paramFile, "%d", &param.nMolecule);
		fgets(skip2, 3, paramFile);
		//read numin
		fgets(skip, 8, paramFile);
		fscanf (paramFile, "%lf", &param.numin);
		fgets(skip2, 3, paramFile);
		//read numax
		fgets(skip, 8, paramFile);
		fscanf (paramFile, "%lf", &param.numax);
		fgets(skip2, 3, paramFile);
		//read dnu
		fgets(skip, 6, paramFile);
		fscanf (paramFile, "%lf", &param.dnu);
		fgets(skip2, 3, paramFile);

	fclose(paramFile);

	//Read console input arguments
	for(int i = 1; i < argc; i += 2){
		if(strcmp(argv[i], "-T") == 0){
			param.T = atof(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-P") == 0){
			param.P = atof(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-m") == 0){
			param.nMolecule = atoi(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-numin") == 0){
			param.numin = atof(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-numax") == 0){
			param.numax = atof(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-dnu") == 0){
			param.dnu = atof(argv[i + 1]);
		}
		else if(strcmp(argv[i], "-dev") == 0){
			param.dev = atoi(argv[i + 1]);
		}
		else{
			printf("Error: Console arguments not valid!\n");
			return 0;
		}

	}
	return 1;
}

int main(int argc, char*argv[]){

	hipError_t error;
	int er;

	int devCount = 0;
	hipGetDeviceCount(&devCount);

	if(devCount == 0){
		printf("Error: No valid cuda device!\n");
		return 0;
	}
	if(devCount == 1) printf("There is %d CUDA Device\n", devCount); 
	else printf("There are %d CUDA Devices\n", devCount); 


	FILE *dataFile;
	FILE *OutFile;
	FILE *Out2File;
	char qFilename[160];
	char paramFilename[160];
	char OutFilename[160];
	char Out2Filename[160];
	sprintf(qFilename, "%s", "q.dat");
	sprintf(paramFilename, "%s", "param.dat");
	sprintf(OutFilename, "%s", "Out.dat");
	sprintf(Out2Filename, "%s", "Out2.dat");

	//Read prameters
	Param param;
	er = read_parameters(param, paramFilename, argc, argv);
	if(er == 0){
		return 0;
	}
	if(param.dev > devCount || param.dev < 0){
		printf("Error: Devive Number is not allowed\n");
		return 0;
	}

	hipSetDevice(param.dev);
	printf("Version: %g\n", VERSION);
	printf("T = %g\nP = %g\nMolecule = %d\nnumin = %g\nnumax = %g\ndnu = %g\n", param.T, param.P, param.nMolecule, param.numin, param.numax, param.dnu);
	printf("Profile = %d\n", PROFILE);
	printf("Using device %d\n", param.dev);

	int Nx = (int)((param.numax - param.numin) / param.dnu);

	Partition part;
	er = ChebCoeff(qFilename, part, param.T);
	if(er == 0){
		return 0;
	}

	Molecule m;
	m.NL = 0;
	m.id = param.nMolecule;	//1 = H2O, 2 = CO, 5 = CO, 6 = CH4
	m.nISO = 0;

	//Initialize the Isotopologue properties for ISO.h
	Init(m);

	if(m.NL == 0){
		printf("Molecule Id is not allowed\n");
		return 0;
	}
	
	
	timeval tt1;			//start time
	timeval tt2;			//end time
	long long times, timems;	//elapsed time in seconds and microseconds

	hipDeviceSynchronize();
	gettimeofday(&tt1, NULL);
	times = 0.0;
	timems = 0.0;
	
	double *nu_h, *nu_d;		//Wavenumber
	double *S_h, *S_d;		//Intensity
	double *A_h, *A_d;		//Einstein A coefficient
	double *delta_h, *delta_d;	//line shift
	double *EL_h, *EL_d;		//lower state energy
	double *alphaL_h, *alphaL_d;	//Lorentz Halfwidth
	double *alphaD_h, *alphaD_d;	//Doppler Halfwidth
	double *n_h, *n_d;		//temperature dependent exponent
	double *gamma_h, *gamma_d;	//pressure shift coefficient
	double *mass_h, *mass_d;
	double *Q_h, *Q_d;		//partition function

	double *K_h, *K_d;
	
	nu_h = (double*)malloc(m.NL * sizeof(double));
	S_h = (double*)malloc(m.NL * sizeof(double));
	A_h = (double*)malloc(m.NL * sizeof(double));
	delta_h = (double*)malloc(m.NL * sizeof(double));
	EL_h = (double*)malloc(m.NL * sizeof(double));
	alphaL_h = (double*)malloc(m.NL * sizeof(double));
	alphaD_h = (double*)malloc(m.NL * sizeof(double));
	n_h = (double*)malloc(m.NL * sizeof(double));
	gamma_h = (double*)malloc(m.NL * sizeof(double));
	mass_h = (double*)malloc(m.NL * sizeof(double));
	Q_h = (double*)malloc(m.NL * sizeof(double));

	hipMalloc((void **) &nu_d, m.NL * sizeof(double));
	hipMalloc((void **) &S_d, m.NL * sizeof(double));
	hipMalloc((void **) &A_d, m.NL * sizeof(double));
	hipMalloc((void **) &delta_d, m.NL * sizeof(double));
	hipMalloc((void **) &EL_d, m.NL * sizeof(double));
	hipMalloc((void **) &alphaL_d, m.NL * sizeof(double));
	hipMalloc((void **) &alphaD_d, m.NL * sizeof(double));
	hipMalloc((void **) &n_d, m.NL * sizeof(double));
	hipMalloc((void **) &gamma_d, m.NL * sizeof(double));
	hipMalloc((void **) &mass_d, m.NL * sizeof(double));
	hipMalloc((void **) &Q_d, m.NL * sizeof(double));


	K_h = (double*)malloc(Nx * sizeof(double));
	hipMalloc((void **) &K_d, Nx * sizeof(double));


	for(int i = 0; i < Nx; ++i){
		K_h[i] = 0.0;
	}

	
	{
		dataFile  = fopen(m.dataFilename, "r");
		if(dataFile == NULL){
			printf("Error: line list file not found\n");
			return 0;
		}
		//read line list file		

		char c1[3];
		//char c2[2];
		char c3[13];
		char c4[11];
		char c5[11];
		char c6[6];
		char c7[6];
		char c8[11];
		char c9[5];
		char c10[9];
		char c11[16];
		char c12[16];
		char c13[16];
		char c14[16];
		char c15[7];
		char c16[13];
		char c17[2];
		char c18[8];
		char c19[8];
		
		char skip[5];

//int count[40];
//for(int cc = 0; cc < 40; ++cc){
//count[cc] = 0;	
//}
		for(int i = 0; i < m.NL; ++i){
		
			fgets(skip, 1, dataFile);
			//fgets(c1, 3, dataFile);
			//fgets(c2, 2, dataFile);
			fgets(c1, 4, dataFile);		//Use combined notation for Id (AFGL and molecule + abundance number
			fgets(c3, 13, dataFile);
			fgets(c4, 11, dataFile);
			fgets(c5, 11, dataFile);
			fgets(c6, 6, dataFile);
			fgets(c7, 6, dataFile);
			fgets(c8, 11, dataFile);
			fgets(c9, 5, dataFile);
			fgets(c10, 9, dataFile);
			fgets(c11, 16, dataFile);
			fgets(c12, 16, dataFile);
			fgets(c13, 16, dataFile);
			fgets(c14, 16, dataFile);
			fgets(c15, 7, dataFile);
			fgets(c16, 13, dataFile);
			fgets(c17, 2, dataFile);
			fgets(c18, 8, dataFile);
			fgets(c19, 8, dataFile);
			fgets(skip, 6, dataFile);
			
			nu_h[i] = strtod(c3, NULL);		
			S_h[i] = strtod(c4, NULL);		
			A_h[i] = strtod(c5, NULL);		
			delta_h[i] = strtod(c9, NULL);
			EL_h[i] = strtod(c8, NULL);		
			
			double gammaAir = strtod(c6, NULL);
			double gammaSelf = strtod(c7, NULL);
			alphaL_h[i] = (1.0 - qALPHA_L) * gammaAir + qALPHA_L * gammaSelf;
			n_h[i] = strtod(c9, NULL);
			alphaD_h[i] = n_h[i];
			
			gamma_h[i] = strtod(c10, NULL);
			int id= std::atoi(c1);
			int idAFGL;
//count[0] += 1;
			//Assign the Isotopologue properties
			for(int j = 0; j < m.nISO; ++j){
				if(id == m.ISO[j].id){
					mass_h[i] = m.ISO[j].m;
					Q_h[i] = m.ISO[j].Q;
					idAFGL = m.ISO[j].AFGL;
				}
			}
			double Q;
			//Assign the Partition function
			for(int j = 0; j < part.n; ++j){
				if(idAFGL == part.id[j]){
					Q = part.Q[j];
				}
			}
			Q_h[i] /= exp(Q);
			
//if(i < 10) printf("%d %d %d %g %g\n", i, id, idAFGL, exp(Q), Q_h[i]);
		}
//for(int cc = 0; cc < 40; ++cc){
//printf("%d %d\n", cc, count[cc]);	
//}
		fclose(dataFile);
	}
	
	
//for(int i = 0; i < m.NL; ++i){
//	printf("%d %g %g\n", i, nu_h[i], S_h[i]);

//}

	hipMemcpy(nu_d, nu_h, m.NL * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(S_d, S_h, m.NL * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(A_d, A_h, m.NL * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(delta_d, delta_h, m.NL * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(EL_d, EL_h, m.NL * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(alphaL_d, alphaL_h, m.NL * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(alphaD_d, alphaD_h, m.NL * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(n_d, n_h, m.NL * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(gamma_d, gamma_h, m.NL * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(mass_d, mass_h, m.NL * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(Q_d, Q_h, m.NL * sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(K_d, K_h, Nx * sizeof(double), hipMemcpyHostToDevice);


	for(int k = 0; k < m.NL; k += nthmax){
		int Nk = min(nthmax, m.NL);
		S_kernel <<< (Nk + 127) / 128, 128 >>> (nu_d, S_d, A_d, EL_d, alphaL_d, alphaD_d, n_d, mass_d, gamma_d, delta_d, Q_d, m.NL, param.T, param.P, k);
	}	

	hipDeviceSynchronize();
	gettimeofday(&tt2, NULL);
	times = (tt2.tv_sec - tt1.tv_sec);
	timems = (tt2.tv_usec - tt1.tv_usec);

	printf("Time before Line_kernel: %g seconds\n", times + timems/1000000.0);
/*
	hipMemcpy(nu_h, nu_d, m.NL * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(S_h, S_d, m.NL * sizeof(double), hipMemcpyDeviceToHost);
	for(int i = 0; i < m.NL; ++i){
		printf("%d %.20g %.20g\n", i, nu_h[i], S_h[i]);

	}
*/
	hipDeviceSynchronize();
	gettimeofday(&tt1, NULL);
	times = 0.0;
	timems = 0.0;

	for(int k = 0; k < Nx; k += nthmax){
		int Nk = min(nthmax, Nx);
		printf("Reached k =  %d, Total = %d\n", k, Nx);
		for(int i = 0; i < m.NL; i += nlmax){
			//This loop reduces the running time of the kernel to a few seconds
			//A longer running time of a single kernel can cause a time out
			Line_kernel <32, nlmax> <<< (Nk + 31) / 32, 32 >>> (nu_d, S_d, alphaL_d, alphaD_d, n_d, K_d, param.dnu, param.numin, Nx, m.NL, i, k);
		}
	}
	hipDeviceSynchronize();
	gettimeofday(&tt2, NULL);
	times = (tt2.tv_sec - tt1.tv_sec);
	timems = (tt2.tv_usec - tt1.tv_usec);

	printf("Time for Line_kernel:    %g seconds\n", times + timems/1000000.0);

	gettimeofday(&tt1, NULL);
	times = 0.0;
	timems = 0.0;

	hipMemcpy(K_h, K_d, Nx * sizeof(double), hipMemcpyDeviceToHost);

	OutFile = fopen(OutFilename, "w");
	for(int j = 0; j < Nx; ++j){
		double x = param.numin + j * param.dnu;
		fprintf(OutFile, "%.20g %.20g\n", x, K_h[j]);
	}
	fclose(OutFile);


	thrust::device_ptr<double> K_dt = thrust::device_pointer_cast(K_d);
	thrust::sort(K_dt, K_dt + Nx);

	hipMemcpy(K_h, K_d, Nx * sizeof(double), hipMemcpyDeviceToHost);

	Out2File = fopen(Out2Filename, "w");
	for(int j = 0; j < Nx; ++j){
		double x = param.numin + j * param.dnu;
		fprintf(Out2File, "%.20g %.20g\n", x, K_h[j]);
	}
	fclose(Out2File);

	hipDeviceSynchronize();
	gettimeofday(&tt2, NULL);
	times = (tt2.tv_sec - tt1.tv_sec);
	timems = (tt2.tv_usec - tt1.tv_usec);

	printf("Time after Line_kernel:  %g seconds\n", times + timems/1000000.0);

	error = hipGetLastError();
	printf("error = %d = %s\n",error, hipGetErrorString(error));

/*
for(int i = 0; i < 1; ++i){

 	double a = 0.0;
/*
	if(i == 0) a = 0.0;
	if(i == 1) a = 0.1;
	if(i == 2) a = 0.3;
	if(i == 3) a = 0.5;
	if(i == 4) a = 0.7;
	if(i == 5) a = 0.9;
	if(i == 6) a = 2.0;
	if(i == 7) a = 5.0;
	if(i == 8) a = 15.0;
* /
/*
	if(i == 0) a = 20.0;
	if(i == 1) a = 40.0;
	if(i == 2) a = 60.0;
	if(i == 3) a = 80.0;
	if(i == 4) a = 100.0;
	if(i == 5) a = 150.0;
	if(i == 6) a = 200.0;
* /	
	a = 200.0;

	double xmax = 100.0;
	Voigt_line_kernel <<< (Nx + 127) / 128, 128 >>> (a, dnu, K_d, Nx, xmax);

	hipMemcpy(K_h, K_d, Nx * sizeof(double), hipMemcpyDeviceToHost);

for(int j = 0; j < Nx; ++j){
	double x = -xmax + j * 2.0 * xmax / ((double)(Nx));
	printf("%.20g %.20g %g\n", x, K_h[j], a);
}
}
*/

	return 0;
}
