#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <sys/time.h>
#include <sys/resource.h>


#include "define.h"
#include "host.h"
#include "ISO.h"
#include "voigt.h"
#include "resample.h"


/*
// runs with biliniar interpolation
// texDescr.filterMode = hipFilterModeLinear;
__global__ void Voigt_texture_kernel(hipTextureObject_t K2dtex, float *K_d, int Nx, int Ny, int Nxtex, int Nytex, size_t pitch){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;


	if(idx < Nx && idy < Ny){
		float x = idx * Nxtex / float(Nx);
		float y = idy * Nytex / float(Ny);
		//float x = idx / float(Nx);
		//float y = idy / float(Ny);
	
		float K = tex2D <float> (K2dtex, x + 0.5f , y + 0.5f);
		float *row = (float *)(((char *)K_d)+(idy*pitch));
    		row[idx] = K;
//if(idy == 0) printf("%d %d %f %f %f\n", idx, idy, x * 10.0f, y * 10.0f, K);

	}
}

// runs with manual biliniar interpolation
// texDescr.filterMode = hipFilterModePoint;
__global__ void Voigt_textureb_kernel(hipTextureObject_t K2dtex, float *K_d, int Nx, int Ny, int Nxtex, int Nytex, size_t pitch){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;


	if(idx < Nx && idy < Ny){
		float x = idx * Nxtex / float(Nx);
		float y = idy * Nytex / float(Ny);
	
		float K00 = tex2D <float> (K2dtex, x, y);
		float K10 = tex2D <float> (K2dtex, x + 1.0f, y);
		float K01 = tex2D <float> (K2dtex, x, y + 1.0f);
		float K11 = tex2D <float> (K2dtex, x + 1.0f, y + 1.0f);

		float xx = (idx % (Nx / Nxtex)) * Nxtex / float(Nx);	
		float yy = (idy % (Ny / Nytex)) * Nytex / float(Ny);	

		float K = (1.0f - xx) * ( 1.0f - yy) * K00 + xx * (1.0f - yy) * K10 + (1.0f - xx) * yy * K01 + xx * yy * K11;

		float *row = (float *)(((char *)K_d)+(idy*pitch));
    		row[idx] = K;
//if(idy == 0) printf("%d %d %f %f | %f %f | %f %f %f %f %f\n", idx, idy, x * 10.0f / Nx, y * 10.0f / Ny, xx, yy, K00, K10, K01, K11, K);

	}
}
// runs with manual biliniar interpolation
// texDescr.filterMode = hipFilterModePoint;
__global__ void Voigt_b_kernel(float *K2d_d, float *K_d, int Nx, int Ny, int Nxtex, int Nytex, size_t pitch){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;


	if(idx < Nx && idy < Ny){
		int x = floor(idx * Nxtex / float(Nx));
		int y = floor(idy * Nytex / float(Ny));
		
		float *row1 = (float *)(((char *)K_d)+(y*pitch)) + x;
		float K00 = *row1;
		float *row2 = (float *)(((char *)K_d)+(y*pitch)) + x + 1;
		float K10 = *row2;
		float *row3 = (float *)(((char *)K_d)+((y + 1)*pitch)) + x;
		float K01 = *row3;
		float *row4 = (float *)(((char *)K_d)+((y + 1)*pitch)) + x + 1;
		float K11 = *row4;

		float xx = (idx % (Nx / Nxtex)) * Nxtex / float(Nx);	
		float yy = (idy % (Ny / Nytex)) * Nytex / float(Ny);	

		float K = (1.0f - xx) * ( 1.0f - yy) * K00 + xx * (1.0f - yy) * K10 + (1.0f - xx) * yy * K01 + xx * yy * K11;

		float *row = (float *)(((char *)K_d)+(idy*pitch));
    		row[idx] = K;
//if(idy == 0) printf("%d %d %f %f | %f %f | %f %f %f %f %f\n", idx, idy, x * 10.0f / Nx, y * 10.0f / Ny, xx, yy, K00, K10, K01, K11, K);

	}
}

//https://stackoverflow.com/questions/34622717/bicubic-interpolation-in-c
__device__ float cubic_hermite(float A, float B, float C, float D, float t){
	float a = -A / 2.0f + (3.0f * B) / 2.0f - (3.0f * C) / 2.0f + D / 2.0f;
	float b =  A - (5.0f * B) / 2.0f + 2.0f * C - D / 2.0f;
	float c = -A / 2.0f + C / 2.0f;
	float d = B;
	float tt = t * t;

	return a * t* tt + b * tt + c * t + d;
}

// runs with manual biliniar interpolation
// texDescr.filterMode = hipFilterModePoint;
__global__ void Voigt_bicubic_kernel(hipTextureObject_t K2dtex, float *K_d, int Nx, int Ny, int Nxtex, int Nytex, size_t pitch){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if(idx > 0 && idy > 0 && idx < Nx - 1&& idy < Ny - 1){
		float x = idx * Nxtex / float(Nx);
		float y = idy * Nytex / float(Ny);
	
		float K00 = tex2D <float> (K2dtex, x - 1.0f, y - 1.0f);
		float K10 = tex2D <float> (K2dtex, x       , y - 1.0f);
		float K20 = tex2D <float> (K2dtex, x + 1.0f, y - 1.0f);
		float K30 = tex2D <float> (K2dtex, x + 2.0f, y - 1.0f);

		float K01 = tex2D <float> (K2dtex, x - 1.0f, y);
		float K11 = tex2D <float> (K2dtex, x       , y);
		float K21 = tex2D <float> (K2dtex, x + 1.0f, y);
		float K31 = tex2D <float> (K2dtex, x + 2.0f, y);

		float K02 = tex2D <float> (K2dtex, x - 1.0f, y + 1.0f);
		float K12 = tex2D <float> (K2dtex, x       , y + 1.0f);
		float K22 = tex2D <float> (K2dtex, x + 1.0f, y + 1.0f);
		float K32 = tex2D <float> (K2dtex, x + 2.0f, y + 1.0f);

		float K03 = tex2D <float> (K2dtex, x - 1.0f, y + 2.0f);
		float K13 = tex2D <float> (K2dtex, x       , y + 2.0f);
		float K23 = tex2D <float> (K2dtex, x + 1.0f, y + 2.0f);
		float K33 = tex2D <float> (K2dtex, x + 2.0f, y + 2.0f);

		float xx = (idx % (Nx / Nxtex)) * Nxtex / float(Nx);	
		float yy = (idy % (Ny / Nytex)) * Nytex / float(Ny);


		float K0 = cubic_hermite(K00, K10, K20, K30, xx);
		float K1 = cubic_hermite(K01, K11, K21, K31, xx);
		float K2 = cubic_hermite(K02, K12, K22, K32, xx);
		float K3 = cubic_hermite(K03, K13, K23, K33, xx);

	
		float K = cubic_hermite(K0, K1, K2, K3, yy);
if(idx == 15 && idy == 15) printf("%d %d %g %g %g %g %g %g %g\n", idx, idy, x, y, K00, K10, K20, K30, K0, K);

		float *row = (float *)(((char *)K_d)+(idy*pitch));
    		row[idx] = K;
//if(idy == 0) printf("%d %d %f %f | %f %f | %f %f %f %f %f\n", idx, idy, x * 10.0f / Nx, y * 10.0f / Ny, xx, yy, K00, K10, K01, K11, K);

	}
}
*/

int main(int argc, char*argv[]){


	hipError_t error;
	int er;

	int devCount = 0;
	hipGetDeviceCount(&devCount);

	if(devCount == 0){
		printf("Error: No valid cuda device!\n");
		return 0;
	}
	if(devCount == 1) printf("There is %d CUDA Device\n", devCount); 
	else printf("There are %d CUDA Devices\n", devCount); 

/*
{

double xMax = 10.0;
double yMax = 10.0;

int Nx = 10000;
int Ny = 10000;

int Nxtex = Nx + 1;
int Nytex = Ny + 1;

int Nxtexf = Nx / 10 + 1;
int Nytexf = Ny / 10 + 1;


double *K2d_h, *K2d_d;
size_t pitch;
//with pitch, the 2d memory is extendend in one dimension to set memory alignment, pitch is the new Nxtex
K2d_h = (double*)malloc( Nxtex * Nytex * sizeof(double));
hipMallocPitch((void **) &K2d_d, &pitch, Nxtex * sizeof(double), Nytex);
//printf("%d %d %lu\n", Nxtex, Nytex, pitch);

{
	double a = (double)(M_PI * sqrt(-1.0 / log(def_TOLF * 0.5)));
	double b = (double)(1.0 / sqrt(M_PI));
	double c = (double)(2.0 * a / M_PI);
	Voigt_2d_kernel <<< dim3((Nxtex + 31) / 32, (Nytex + 31) / 32), dim3(32, 32, 1) >>> (a, b, c, K2d_d, Nxtex, Nytex, pitch, xMax, xMax);
	hipMemcpy2D(K2d_h, Nxtex * sizeof(double), K2d_d, pitch, Nxtex * sizeof(double), Nytex, hipMemcpyDeviceToHost);
}
/ *
for(int i = 0; i < Nxtex - 1; ++i){
	for(int j = 0; j < Nytex - 1; ++j){
		double x = i * xMax / double(Nxtex);
		double y = j * yMax / double(Nytex);
		if( x < xMax && y < yMax){
			printf("%g %g %.15g\n", x, y, K2d_h[j * Nxtex + i]);
		}
	}
}
* /

float *K2df_h, *K2df_d;
size_t pitchf;
//with pitchf, the 2d memory is extendend in one dimension to set memory alignment, pitchf is the new Nxtexf
K2df_h = (float*)malloc( Nxtexf * Nytexf * sizeof(float));
hipMallocPitch((void **) &K2df_d, &pitchf, Nxtexf * sizeof(float), Nytexf);
//printf("%d %d %lu\n", Nxtexf, Nytexf, pitchf);

{
	float a = (float)(M_PI * sqrt(-1.0f / log(def_TOLF * 0.5f)));
	float b = (float)(1.0f / sqrt(M_PI));
	float c = (float)(2.0f * a / M_PI);
	Voigt_2df_kernel <<< dim3((Nxtexf + 31) / 32, (Nytexf + 31) / 32), dim3(32, 32, 1) >>> (a, b, c, K2df_d, Nxtexf, Nytexf, pitchf, xMax, xMax);
	hipMemcpy2D(K2df_h, Nxtexf * sizeof(float), K2df_d, pitchf, Nxtexf * sizeof(float), Nytexf, hipMemcpyDeviceToHost);
}
/ *
for(int i = 0; i < Nxtexf - 1; ++i){
	for(int j = 0; j < Nytexf -1; ++j){
		float x = i * xMax / float(Nxtexf - 1);
		float y = j * yMax / float(Nytexf - 1);
		if( x < xMax && y < yMax){
			printf("%g %g %.15g\n", x, y, K2df_h[j * Nxtexf + i]);
		}
	}
}

return 0;
* /
//https://stackoverflow.com/questions/41749024/edit-cuda-texture-object
hipTextureObject_t K2dtex;

hipResourceDesc resDescr;
memset(&resDescr, 0, sizeof(hipResourceDesc));
resDescr.resType = hipResourceTypePitch2D;
resDescr.res.pitch2D.desc = hipCreateChannelDesc<float>();
resDescr.res.pitch2D.devPtr = K2df_d;
resDescr.res.pitch2D.height = Nytexf;
resDescr.res.pitch2D.pitchInBytes = pitchf;
resDescr.res.pitch2D.width = Nxtexf;


hipTextureDesc  texDescr;
memset(&texDescr, 0, sizeof(hipTextureDesc));
texDescr.normalizedCoords = 0;
//texDescr.filterMode = hipFilterModeLinear;
texDescr.filterMode = hipFilterModePoint;
texDescr.addressMode[0] = hipAddressModeClamp;
texDescr.addressMode[1] = hipAddressModeClamp;
texDescr.addressMode[2] = hipAddressModeClamp;
texDescr.readMode = hipReadModeElementType;

hipCreateTextureObject(&K2dtex, &resDescr, &texDescr, NULL);



float *K_h, *K_d;
K_h = (float*)malloc( Nx * Ny * sizeof(float));
//with pitch, the 2d memory is extendend in one dimension to set memory alignment, pitch is the new Nx
hipMallocPitch((void **) &K_d, &pitch, Nx * sizeof(float), Ny);


for(int t = 0; t < 1; ++t){
	//Voigt_texture_kernel <<< dim3((Nx + 31) / 32, (Ny + 31) / 32), dim3(32, 32, 1) >>> (K2dtex, K_d, Nx, Ny, Nxtexf - 1, Nytexf - 1, pitch);
	//Voigt_textureb_kernel <<< dim3((Nx + 31) / 32, (Ny + 31) / 32), dim3(32, 32, 1) >>> (K2dtex, K_d, Nx, Ny, Nxtexf -1, Nytexf - 1, pitch);
	//Voigt_b_kernel <<< dim3((Nx + 31) / 32, (Ny + 31) / 32), dim3(32, 32, 1) >>> (K2d_d, K_d, Nx, Ny, Nxtex - 1, Nytex - 1, pitch);
	Voigt_bicubic_kernel <<< dim3((Nx + 31) / 32, (Ny + 31) / 32), dim3(32, 32, 1) >>> (K2dtex, K_d, Nx, Ny, Nxtexf - 1, Nytexf - 1, pitch);
}

hipMemcpy2D(K_h, Nx * sizeof(float), K_d, pitch, Nx * sizeof(float), Ny, hipMemcpyDeviceToHost);
hipDeviceSynchronize();

for(int i = 0; i < Nx; ++i){
	for(int j = 0; j < Ny; ++j){
		double x = i * xMax / double(Nx);
		double y = j * yMax / double(Ny);
		if( x < xMax && y < yMax){
			double diff = fabs(K2d_h[j * Nxtex + i] - K_h[j * Nx + i]);
			if(diff > 5.0e-7){
				printf("%g %g %.15g %.15g %.15g\n", x, y, K2d_h[j * Nxtex + i], K_h[j * Nx + i], diff);
			}
		}
	}
}
return 0;
}
*/
	char qFilename[15][160];	//for maximal 15 isotopologues
	char paramFilename[160];
	sprintf(paramFilename, "%s", "param.dat");

	//Read prameters
	Param param;
	param.dev = 0;
	param.useIndividualBins = 0;
	param.useOutputEdges = 0;
	param.nedges = 0;
	param.nP = 1;
	param.usePFile = 0;
	param.useIndividualX = 0;
	param.useCia = 0;

	param.T = 0.0;
	param.P = 0.0;
	param.useHITEMP = 0;
	param.nMolecule = 0;
	param.numin = 0.0;
	param.numax = 0.0;
	param.dnu = 0.0;
	param.Nxb = 0;
	param.cutMode = 0;
	param.cut = 0.0;
	param.doResampling = 0;
	param.nC = 0;
	param.doTransmission = 0;
	param.nTr = 0;
	param.dTr = 0.0;
	param.doStoreFullK = 0;
	param.doStoreK = 0;
	param.nbins = 0;
	param.kmin = 0.0;
	param.qalphaL = def_qALPHA_L;
	param.doMean = 0;
	param.units = 0;	
	param.replaceFiles = 0;
	param.RLOW = 0;
	param.profile = def_PROFILE;
	
	er = read_parameters(param, paramFilename, argc, argv);
	if(er == 0){
		return 0;
	}
	if(param.dev >= devCount || param.dev < 0){
		printf("Error: Device Number is not allowed\n");
		return 0;
	}
	if(param.useIndividualX == 1 && param.RLOW == 1){
		printf("Error: bins file and RLOW  not allowed\n");
		return 0;
	}

	char filemode[16];
	if(param.replaceFiles == 0){
		sprintf(filemode, "a");
	}
	else{
		sprintf(filemode, "w");
	}

	FILE *InfoFile;
	char InfoFilename[160];
	sprintf(InfoFilename, "Info_%s.dat", param.name);
	InfoFile = fopen(InfoFilename, filemode);

	int runtimeVersion;
	int driverVersion;

	hipRuntimeGetVersion(&runtimeVersion);
	hipDriverGetVersion(&driverVersion);

	hipSetDevice(param.dev);
	hipDeviceProp_t devProp;
	for(int i = 0; i < 2; ++i){
		FILE *infofile;
		if(i == 0) infofile = InfoFile;
		if(i == 1) infofile = stdout;

		for(int j = 0; j < devCount; ++j){
			hipGetDeviceProperties(&devProp, j);
			fprintf(infofile,"Name:%s, Major:%d, Minor:%d, Max threads per Block:%d, Max x dim:%d\n, #Multiprocessors:%d, Clock Rate:%d, Memory Clock Rate:%d, Global Memory:%lu, Shared memory per block: %lu\n",
				devProp.name, devProp.major, devProp.minor, devProp.maxThreadsPerBlock, devProp.maxThreadsDim[0],
				devProp.multiProcessorCount,  devProp.clockRate, devProp.memoryClockRate, devProp.totalGlobalMem, devProp.sharedMemPerBlock);

		}
	}
	if(param.Nxb != 0){
		param.useIndividualX = 1;
	}

	//If the bin file is used, store the boundaries of the bins
	double *binBoundaries_h, *binBoundaries_d;
	binBoundaries_h = (double*)malloc((param.nbins + 1) * sizeof(double));
	hipMalloc((void **) &binBoundaries_d, (param.nbins + 1) * sizeof(double));
	if(param.useIndividualBins == 1){
		er = readBinFile(param, binBoundaries_h);
		if(er == 0) return 0;
		param.numin = binBoundaries_h[0];
		param.numax = binBoundaries_h[param.nbins];

		if(param.doResampling > 0){
			printf("Error: The resampling function is not supported for the bin-file option\n");
			return 0;
		}
		if(param.doTransmission > 0){
			printf("Error: The transmission function is not supported for the bin-file option\n");
			return 0;
		}
	}
	else{
		for(int i = 0; i < param.nbins; ++i){
			binBoundaries_h[i] = param.numin + i * (param.numax - param.numin) / ((double)(param.nbins));
		}
		binBoundaries_h[param.nbins] = param.numax;
	}
	hipMemcpy(binBoundaries_d, binBoundaries_h, (param.nbins + 1) * sizeof(double), hipMemcpyHostToDevice);

//for(int i = 0; i < param.nbins + 1; ++i){
//	printf("binboundaries %d %g\n", i, binBoundaries_h[i]);
//}	

	int Nx;
	if(param.useIndividualX == 0){
		Nx = (int)((param.numax - param.numin) / param.dnu + 0.5); //+ 0.5 to round correctly between double and int
		if((param.numax - param.numin) / param.dnu + 0.5 >= 2147483647){
			printf("Error: Nx too large, integer overflow. %d %g\n", Nx, (param.numax - param.numin) / param.dnu);
			return 0;
		}
printf("%g %g %g %g\n", param.numax, param.numin, param.dnu, (param.numax - param.numin) / param.dnu + 0.5);
		param.Nxb = Nx / param.nbins;
		if(Nx % param.nbins != 0){
			printf("Error: range cannot be divided evenly in bins. %d %d %g\n", Nx, param.nbins,  Nx / ((double)(param.nbins)));
			return 0;
		}
	}
	else{
		Nx = param.nbins * param.Nxb;
		if(param.nbins * param.Nxb >= 2147483647){
			printf("Error: Nx too large, integer overflow. %d %g\n", Nx, (double)(param.nbins) * (double)(param.Nxb));
			return 0;
		}
		if(param.doResampling > 0){
			printf("Error: The resampling function is not supported for unequal spacing option\n");
			return 0;
		}
		if(param.doTransmission > 0){
			printf("Error: The transmission function is not supported for unequal spacing option\n");
			return 0;
		}
	}


	int Nx1 = (Nx + 9) / 10;

	//If the output edges file is used store the edges
	double *outputEdges_h;
	if(param.useOutputEdges == 1){
		outputEdges_h = (double*)malloc((param.nedges + 1) * sizeof(double));
		er = readEdgesFile(param, outputEdges_h);
		if(er == 0) return 0;
	}
	else{
		outputEdges_h = NULL;
	}
	//Allocate P array 
	double *P_h;
	P_h = (double*)malloc((param.nP) * sizeof(double));
	P_h[0] = param.P;
	if(param.usePFile == 1){
		er = readPFile(param, P_h);
		if(er == 0) return 0;
	}

	double time[9];
	for(int i = 0; i < 9; ++i){
		time[i] = 0.0;
	}

	//Allocate Molecule properties
	Molecule m;
	m.NL[0] = 0;
	m.id = param.nMolecule;	//1 = H2O, 2 = CO, 5 = CO, 6 = CH4
	m.nISO = 0;
	m.defaultL = 0.0;
	m.defaultn = 0.0;
	//Initialize the Isotopologue properties for ISO.h
	Init(m, param, qFilename);

	for(int i = 0; i < 2; ++i){
		FILE *infofile;
		if(i == 0) infofile = InfoFile;
		if(i == 1) infofile = stdout;
		fprintf(infofile, "\nVersion: %g\n", VERSION);
		fprintf(infofile, "Using device %d\n\n", param.dev);
		fprintf(infofile, "Runtime Version %d\n", runtimeVersion);
		fprintf(infofile, "Driver Version %d\n", driverVersion);

		if(param.Nxb < param.nC && i == 0){
			printf("Number of points per bin smaller than the number of Chebyshev coefficients: Changed nC to %d\n", param.Nxb);
			fprintf(infofile, "Number of points per bin smaller than the number of Chebyshev coefficients: Changed nC to %d\n", param.Nxb);
			param.nC = param.Nxb;
		}
		fprintf(infofile, "name = %s\n", param.name);
		fprintf(infofile, "T = %g\n", param.T);
		if(param.usePFile == 0){
			fprintf(infofile, "P = %g\n", P_h[0]);
		}
		else{
			fprintf(infofile, "P in file: %s\n", param.PFilename);
		}
		fprintf(infofile, "useHITEMP = %d\n", param.useHITEMP);
		fprintf(infofile, "Molecule = %d\n", param.nMolecule);
		fprintf(infofile, "cia System = %s\n", param.ciaSystem);
		fprintf(infofile, "pathToData = %s\n", param.path);
		fprintf(infofile, "numin = %g\n", param.numin);
		fprintf(infofile, "numax = %g\n", param.numax);
		fprintf(infofile, "dnu = %g\n", param.dnu);
		fprintf(infofile, "Nnu per bin = %d\n", param.Nxb);
		fprintf(infofile, "Number of points: %d\n", Nx);
		fprintf(infofile, "cutMode = %d\n", param.cutMode);
		fprintf(infofile, "cut = %g\n", param.cut);
		fprintf(infofile, "doResampling = %d\n", param.doResampling);
		fprintf(infofile, "nC = %d\n", param.nC);
		fprintf(infofile, "doTransmission = %d\n", param.doTransmission);
		fprintf(infofile, "nTr = %d\n", param.nTr);
		fprintf(infofile, "dTr =  %g\n", param.dTr);
		fprintf(infofile, "doStoreFullK = %d\n", param.doStoreFullK);
		fprintf(infofile, "pathToK = %s\n", param.pathK);
		fprintf(infofile, "dostoreK = %d\n", param.doStoreK);
		fprintf(infofile, "nbins = %d\n", param.nbins);
		if(param.useIndividualBins == 1){
			fprintf(infofile, "use Individual bins: %s\n", param.bins);
		}
		fprintf(infofile, "kmin = %g\n", param.kmin);
		fprintf(infofile, "qalphaL = %g\n", param.qalphaL);
		fprintf(infofile, "doMean = %d\n", param.doMean);
		fprintf(infofile, "Units = %d\n", param.units);
		fprintf(infofile, "Replace files = %d\n", param.replaceFiles);
		fprintf(infofile, "default L = %g\n", m.defaultL);
		fprintf(infofile, "default n = %g\n", m.defaultn);
		fprintf(infofile, "RLOW = %d\n", param.RLOW);
		fprintf(infofile, "profile = %d\n", param.profile);
		fprintf(infofile, "def_TOL = %g\n", def_TOL);
		fprintf(infofile, "def_TOLf = %g\n", def_TOLF);
		fprintf(infofile, "def_nthmax = %d\n", def_nthmax);
		fprintf(infofile, "def_nlmax = %d\n", def_nlmax);
		fprintf(infofile, "def_maxlines = %d\n", def_maxlines);
		fprintf(infofile, "def_maxfiles = %d\n", def_maxfiles);
		fprintf(infofile, "def_NmaxSample = %d\n", def_NmaxSample);
		fprintf(infofile, "def_NXLOW = %d\n", def_NXLOW);
		if(param.useOutputEdges == 1){
			fprintf(infofile, "use output edges: %s\n", param.edges);
		}

	}
	fclose(InfoFile);
	
	//Read partition function
	Partition part;
	er = readPartition(param, param.nMolecule, qFilename, part, param.T, m);
	if(er == 0){
		return 0;
	}

	if(param.useHITEMP == 2 && m.defaultL == 0.0){
		printf("Molecule Id is not allowed for ExoMol\n");
		return 0;
	}

	//compute the mean mass
	m.meanMass = 0.0;
	for(int i = 0; i < m.nISO; ++i){
		m.meanMass += m.ISO[i].Ab * m.ISO[i].m; //mean Molar Mass (g)
	}
//printf("mean mass %g\n", m.meanMass);
	double unitScale = 1.0;

	//Set cia System properties
	ciaSystem cia;
	if(param.useCia == 1){
		er = InitCia(m, cia, param);
		if(er == 0) return 0;
	}

	if(param.useCia == 1 && param.nMolecule != 0){
		printf("Error, not allowed to use a cia system with a molecule\n");
		return 0;
	}


	if(param.units == 1){
		unitScale = 1.0 / def_NA * m.meanMass;
		if(param.useCia == 1 && param.nMolecule == 0){
			unitScale = 1.0 / def_NA * cia.mass1;
		}
		param.kmin /= unitScale;
	}	

	timeval tt1;			//start time
	timeval tt2;			//end time
	long long times, timems;	//elapsed time in seconds and microseconds
	hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
	float milliseconds;

	hipDeviceSynchronize();

	Line L;

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != 0){
		printf("Initial error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}

	//Allocate memory for Line properties
	if(param.useHITEMP < 2){
		Alloc_Line(L, m);
	}
	else{
		Alloc2_Line(L, m);
	}
	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != 0){
		printf("Line alloc error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}
	double *K_h, *K_d;
	double *K1_d;
	double *Kc_d;
	double *x_h, *x_d;
	int *binKey_d;
	int *binIndex_h, *binIndex_d;
	K_h = (double*)malloc(Nx * sizeof(double));
	x_h = (double*)malloc(Nx * sizeof(double));
	binIndex_h = (int*)malloc((param.nbins + 2) * sizeof(int));
	hipMalloc((void **) &K_d, param.nP * Nx * sizeof(double));
	if(param.RLOW == 1){
		hipMalloc((void **) &K1_d, Nx1 * sizeof(double));
		hipMalloc((void **) &Kc_d, Nx * sizeof(double));
	}
	else{
		K1_d = NULL;
		Kc_d = NULL;
	}
	hipMalloc((void **) &x_d, Nx * sizeof(double));
	hipMalloc((void **) &binKey_d, Nx * sizeof(int));
	hipMalloc((void **) &binIndex_d, (param.nbins + 2) * sizeof(int));

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != 0){
		printf("K alloc error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}
	for(int k = 0; k < param.nP * Nx; k += def_nthmax){
		int Nk = min(def_nthmax, param.nP * Nx - k);
		InitialK_kernel <<< (Nk + 511) / 512, 512 >>> (K_d, param.nP * Nx, param.kmin, k);
	}
	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != 0){
		printf("bin1 error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}
	for(int k = 0; k < Nx; k += def_nthmax){
		int Nk = min(def_nthmax, Nx - k);
		setX_kernel <<< (Nk + 511) / 512, 512 >>> (x_d, Nx, param.numin, param.dnu, param.Nxb, param.useIndividualX, binBoundaries_d, k);
	}
	hipMemcpy(x_h, x_d, Nx * sizeof(double), hipMemcpyDeviceToHost);
	for(int k = 0; k < Nx; k += def_nthmax){
		int Nk = min(def_nthmax, Nx - k);
		binKey_kernel <<< (Nk + 511) / 512, 512 >>> (binKey_d, Nx, param.Nxb, binBoundaries_d, param.nbins, param.numax, x_d, param.useIndividualX, k);
	}
	for(int k = 0; k < Nx; k += def_nthmax){
		int Nk = min(def_nthmax, Nx - k);
		binIndex_kernel <<< (Nk + 511) / 512, 512 >>> (binKey_d, binIndex_d, Nx, param.nbins, k);
	}
	hipMemcpy(binIndex_h, binIndex_d, (param.nbins + 2) * sizeof(int), hipMemcpyDeviceToHost);


	/*			
	int *binKey_h; 	//only needed to check the key
	binKey_h = (int*)malloc(Nx * sizeof(int));
	hipMemcpy(binKey_h, binKey_d, Nx * sizeof(int), hipMemcpyDeviceToHost);
	for(int i = 0; i < Nx; ++i){
		int bin = binKey_h[i];
		printf("%d %.10g %d %d %d\n", i, x_h[i], bin, binIndex_h[bin], binIndex_h[bin + 1]);
	}
	*/

	const int ntL = 64;	//number of threads in Line kernel
	int nLimits = (Nx + ntL - 1) / ntL;
	int2 *Limits_d;

	hipMalloc((void **) &Limits_d, nLimits * sizeof(int2));
	int *MaxLimits_h, *MaxLimits_d;
	MaxLimits_h = (int*)malloc(sizeof(int));
	hipMalloc((void **) &MaxLimits_d, sizeof(int));

	if(param.useCia == 1){
		for(int iP = 0; iP < param.nP; ++iP){
			readCiaFile(param, cia, x_h, K_h, Nx, param.T, P_h[iP], m.meanMass);
			hipMemcpy(K_d + iP * Nx, K_h, Nx * sizeof(double), hipMemcpyHostToDevice);
		}
	}

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != 0){
		printf("Alloc error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}

	if(param.nMolecule > 0 && param.doStoreFullK >= 0){
		double *nuP;
		double *ialphaDP;
		double *vyP;
		nuP = NULL;
		ialphaDP = NULL;
		vyP = NULL;
		if(Nx > def_NXLOW){
			int n = min(def_maxlines, m.NLmax);
			nuP = (double*)malloc(n * sizeof(double));
			ialphaDP = (double*)malloc(n * sizeof(double));
			vyP = (double*)malloc(n * sizeof(double));
		}

		//**************************************
		//Starting the loop around the datafiles
		//**************************************
		int fi0 = m.nFiles;
		int fi1 = 0;


		if(param.cutMode == 0 && param.cut){
			for(int fi = 0; fi < m.nFiles; ++fi){
				if(m.fileLimit[fi] - param.cut <= param.numax) fi1 = fi + 1;
				else break;
			}
			for(int fi = m.nFiles - 1; fi >= 0; --fi){
				if(m.fileLimit[fi + 1] + param.cut >= param.numin) fi0 = fi;
				else break;
			}
		}
		else{
			fi0 = 0;
			fi1 = m.nFiles;
		}

		printf("File range %d to %d\n", fi0 + 1, fi1);

		time[0] = 0.0;
		time[1] = 0.0;
		time[2] = 0.0;
	
		int fi;
		for(fi = fi0; fi < fi1; ++fi){


			FILE *dataFile;
			char dataFilename[160];
			sprintf(dataFilename, "%sbin", m.dataFilename[fi]);
			dataFile  = fopen(dataFilename, "rb");

			if(dataFile == NULL){
				printf("Error: line list file not found %s\n", dataFilename);
				return 0;
			}

			printf("Reading file %d of %d\n", fi + 1, fi1);
			printf("Number of lines: %lld\n", m.NL[fi]);

			for(long long int iL = 0LL; iL < m.NL[fi]; iL += def_maxlines){
				int NL = min(def_maxlines, m.NL[fi] - iL);
				printf("Reading Line file part %d of %d with %d lines\n", (iL + def_maxlines - 1) / def_maxlines + 1, (m.NL[fi] + def_maxlines - 1) / def_maxlines, NL);

				double timeOld = time[0];
				if(iL == 0) time[0] = 0.0;
				gettimeofday(&tt1, NULL);
				//**************************
				//Read the Line list	
				//**************************
				if(param.useHITEMP < 2){
					er = readFile(param, m, part, L, param.qalphaL, NL, dataFile);
				}
				else{
					er = readFileExomol(param, m, part, L, NL, dataFile);
				}
				if(er == 0){
					return 0;
				}
				gettimeofday(&tt2, NULL);
				times = (tt2.tv_sec - tt1.tv_sec);
				timems = (tt2.tv_usec - tt1.tv_usec);
				time[0] += times + timems/1000000.0;
				
				printf("Time for input:        %g seconds\n", time[0]);

				if(iL > 0 || fi > fi0){
					//read data before synchronization
					if(m.NL[fi - 1] > 0){
						hipEventSynchronize(stop);
						hipEventElapsedTime(&milliseconds, start, stop);

						time[2] += milliseconds * 0.001;
					}
					printf("Time for K(x):         %g seconds\n", time[2]);
		
					hipDeviceSynchronize();
					error = hipGetLastError();
					if(error != 0){
						printf("Ka error = %d = %s\n",error, hipGetErrorString(error));
						return 0;
					}
					if(iL == 0){
						InfoFile = fopen(InfoFilename, "a");
						fprintf(InfoFile,"File %d of %d\n", fi, fi1);
						fprintf(InfoFile,"Number of lines: %lld\n", m.NL[fi - 1]);
						fprintf(InfoFile,"Time for input:        %g seconds\n", timeOld);
						fprintf(InfoFile,"Time for Lines:        %g seconds\n", time[1]);
						fprintf(InfoFile,"Time for K(x):         %g seconds\n", time[2]);
						fclose(InfoFile);
						time[1] = 0.0;
						time[2] = 0.0;
					}
				}

				hipEventRecord(start);

				//start the loop around the Pressure values. only 1 iteration if no Pressure file is given
				for(int iP = 0; iP < param.nP; ++iP){

					//Copy Line data to the device

					if(param.RLOW == 1){
						hipMemset(K1_d, 0, Nx1 * sizeof(double));	
						hipMemset(Kc_d, 0, Nx * sizeof(double));	
					}

					if(param.useHITEMP < 2){
						Copy_Line(L, m, NL);
					}
					else{
						Copy2_Line(L, m, NL);
					}
					//************************

					//***************************
					//Compute Line properties
					//***************************
					if(param.useHITEMP < 2){
						for(int k = 0; k < NL; k += def_nthmax){
							int Nk = min(def_nthmax, NL);
							if(Nk > 0) S2_kernel <<< (Nk + 127) / 128, 128 >>> (L.nu_d, L.S_d, L.Sf_d, L.A_d, L.vy_d, L.vyf_d, L.ialphaD_d, L.n_d, L.delta_d, L.EL_d, L.ID_d, L.va_d, L.vb_d, L.vcut2_d, L.S1_d, L.S1f_d, NL, param.numin, param.dnu, param.cut, param.cutMode, param.profile, param.useIndividualX, param.T, P_h[iP], k);
						}	
			/* // *************
						//uncoment this only when no Pressure file is given
						//print number of lines per bin
						hipMemcpy(L.nu_h, L.nu_d, NL * sizeof(double), hipMemcpyDeviceToHost);
						int nLb[param.nbins];
						for(int i = 0; i < param.nbins; ++i){
							nLb[i] = 0;
						}
						double binWidth = (param.numax - param.numin) / ((double)(param.nbins));
						printf("%g\n", binWidth);
						for(int i = 0; i < NL; ++i){
							int b = int(L.nu_h[i] / binWidth);
							nLb[b] += 1;
						}
						for(int i = 0; i < param.nbins; ++i){
							printf("%d, ", nLb[i]);
						}
						printf("\n");
			 
			*/
					}
					else{
						for(int k = 0; k < NL; k += def_nthmax){
							int Nk = min(def_nthmax, NL);
							if(Nk > 0) Sf_kernel <<< (Nk + 127) / 128, 128 >>> (L.nu_d, L.S_d, L.Sf_d, L.A_d, L.vy_d, L.vyf_d, L.ialphaD_d, L.n_d, L.EL_d, L.S1_d, L.S1f_d, L.va_d, L.vb_d, L.vcut2_d, NL, param.numin, param.dnu, param.cut, param.cutMode, param.profile, param.useIndividualX, param.T, P_h[iP], k);
						}
					}
					//Sort the data along nu
					thrust::device_ptr<double> nu_dt = thrust::device_pointer_cast(L.nu_d);
					thrust::device_ptr<int> ID_dt = thrust::device_pointer_cast(L.ID_d);

					thrust::sort_by_key(nu_dt, nu_dt + NL, ID_dt);

					//Destroy Q_d to sort S_d vy_d and ialphaD_d
					int Nk = min(def_nthmax, NL);
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Copy_kernel <<< (Nk + 127) / 128, 128 >>> (L.S_d, L.Q_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Sort_kernel <<< (Nk + 127) / 128, 128 >>> (L.Q_d, L.S_d, L.ID_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Copyf_kernel <<< (Nk + 127) / 128, 128 >>> (L.Sf_d, L.Q_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Sortf_kernel <<< (Nk + 127) / 128, 128 >>> (L.Q_d, L.Sf_d, L.ID_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Copy_kernel <<< (Nk + 127) / 128, 128 >>> (L.vy_d, L.Q_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Sort_kernel <<< (Nk + 127) / 128, 128 >>> (L.Q_d, L.vy_d, L.ID_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Copyf_kernel <<< (Nk + 127) / 128, 128 >>> (L.vyf_d, L.Q_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Sortf_kernel <<< (Nk + 127) / 128, 128 >>> (L.Q_d, L.vyf_d, L.ID_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Copy_kernel <<< (Nk + 127) / 128, 128 >>> (L.ialphaD_d, L.Q_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Sort_kernel <<< (Nk + 127) / 128, 128 >>> (L.Q_d, L.ialphaD_d, L.ID_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Copyf_kernel <<< (Nk + 127) / 128, 128 >>> (L.va_d, L.Q_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Sortf_kernel <<< (Nk + 127) / 128, 128 >>> (L.Q_d, L.va_d, L.ID_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Copyf_kernel <<< (Nk + 127) / 128, 128 >>> (L.vb_d, L.Q_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Sortf_kernel <<< (Nk + 127) / 128, 128 >>> (L.Q_d, L.vb_d, L.ID_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Copyf_kernel <<< (Nk + 127) / 128, 128 >>> (L.vcut2_d, L.Q_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Sortf_kernel <<< (Nk + 127) / 128, 128 >>> (L.Q_d, L.vcut2_d, L.ID_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Copy_kernel <<< (Nk + 127) / 128, 128 >>> (L.S1_d, L.Q_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Sort_kernel <<< (Nk + 127) / 128, 128 >>> (L.Q_d, L.S1_d, L.ID_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Copyf_kernel <<< (Nk + 127) / 128, 128 >>> (L.S1f_d, L.Q_d, NL, k);
					}
					for(int k = 0; k < NL; k += def_nthmax){
						if(Nk > 0) Sortf_kernel <<< (Nk + 127) / 128, 128 >>> (L.Q_d, L.S1f_d, L.ID_d, NL, k);
					}
					if(Nx > def_NXLOW){
						hipMemcpy(nuP, L.nu_d, NL * sizeof(double), hipMemcpyDeviceToHost);
						hipMemcpy(ialphaDP, L.ialphaD_d, NL * sizeof(double), hipMemcpyDeviceToHost);
						hipMemcpy(vyP, L.vy_d, NL * sizeof(double), hipMemcpyDeviceToHost);
					}
					//********************************

					hipDeviceSynchronize();
					error = hipGetLastError();
					if(error != 0){
						printf("Sort error = %d = %s\n",error, hipGetErrorString(error));
						return 0;
					}
					if(Nx <= def_NXLOW){
						//********************************
						//Determine which lines the block in the Line kernel has to read
						//********************************
						hipMemset(MaxLimits_d, 0, sizeof(int));

						setLimits_kernel <<< (nLimits + 255) / 256, 256 >>> (Limits_d, nLimits, NL, param.cut);
						if(param.cut != 0.0){
							if(NL > 0) Cutoff_kernel <<< (NL + 255) / 256 , 256 >>> (L.nu_d, L.ID_d, Limits_d, L.vy_d, L.ialphaD_d, ntL, param.numin, param.dnu, NL, nLimits, param.cut, param.cutMode, Nx, x_d, param.useIndividualX);
							MaxLimits_kernel <<< (nLimits + 255) / 256, 256 >>> (Limits_d, MaxLimits_d, nLimits, NL);
							hipMemcpy(MaxLimits_h, MaxLimits_d, sizeof(int), hipMemcpyDeviceToHost);
						}
						else MaxLimits_h[0] = NL;
						printf("Maximum number of Line Blocks %d\n", MaxLimits_h[0]);
						
	/*					//print Limits
						int2 *Limits_h;
						Limits_h = (int2*)malloc(nLimits * sizeof(int2));
						hipMemcpy(Limits_h, Limits_d, nLimits * sizeof(int2), hipMemcpyDeviceToHost);
						FILE *LimitsFile;
						char LimitsFilename[160];
						sprintf(LimitsFilename, "Limits_%s.dat", param.name);
						if(fi == 0){
							LimitsFile = fopen(LimitsFilename, filemode);
						}
						else{
							LimitsFile = fopen(LimitsFilename, "a");
						}

						for(int i = 0; i < nLimits; ++i){
							fprintf(LimitsFile,"%d %d %d %d\n", fi, i, Limits_h[i].x, Limits_h[i].y);
						}
						fclose(LimitsFile);
						free(Limits_h);
	*/	 				
						//*********************************************
					}

					hipEventRecord(stop);
					hipEventSynchronize(stop);
					error = hipGetLastError();
					if(error != 0){
						printf("Line error = %d = %s\n",error, hipGetErrorString(error));
						return 0;
					}
					hipEventElapsedTime(&milliseconds, start, stop);

					time[1] += milliseconds * 0.001;

					if(iP == param.nP - 1){
						printf("Time for Lines:        %g seconds\n", time[1]);
					}

					hipEventRecord(start);

					//***********************************
					//Compute the opacity function K(x)
					//************************************
					double cut = param.cut;
					if(cut == 0.0) cut = 1.0e30;
					if(Nx <= def_NXLOW){
						float a = (float)(M_PI * sqrt(-1.0 / log(def_TOLF * 0.5)));
						float b = (float)(1.0 / sqrt(M_PI));
						float c = (float)(2.0 * a / M_PI);
						for(int k = 0; k < Nx; k += def_nthmax){
							int Nk = min(def_nthmax, Nx - k);
							for(int i = 0; i < MaxLimits_h[0]; i += def_nlmax){
if(i % (1000 * def_nlmax) == 0){
printf("%d %d %d %d\n", MaxLimits_h[0], def_nlmax, k, ntL);
}
								int nl = min(MaxLimits_h[0] - i, def_nlmax);
								//This loop reduces the running time of the kernel to a few seconds
								//A longer running time of a single kernel can cause a time out
								Line_kernel < ntL > <<< (Nk + ntL - 1) / ntL, ntL >>> (L.Sf_d, L.S1f_d, L.vyf_d, L.va_d, L.vb_d, L.vcut2_d, K_d + iP * Nx, x_d, Nx, NL, Limits_d, nl, i, k, param.useIndividualX, param.Nxb, binBoundaries_d, a, b, c, param.profile);
							}
						}
					}
					else{ // end Nx <= def_NXLOW
						if(param.RLOW == 0){
							const int nl = 512;
							for(int il = 0; il < NL; il += nl){ //loop over lines
								int ii11 = 0;
								int ii00 = Nx;
								if(param.useIndividualX == 0){
									for(int iil = 0; iil < nl; ++iil){
										if(il + iil < NL){
											int Inu = (int)((nuP[il + iil] - param.numin) / param.dnu);
											int ii0 = Inu - (int)(cut / param.dnu) - 1;
											int ii1 = Inu + (int)(cut / param.dnu) + 2;

	//if(iil % 10000 == 0) printf("%d %.30g %d %d %d\n", il + iil, nuP[il + iil], Inu, ii0, ii1);

											ii11 = max(ii11, ii1);
											ii00 = min(ii00, ii0);
										}
									}
								}
								else{
									double nu00 = param.numax;
									double nu11 = param.numin;
									for(int iil = 0; iil < nl; ++iil){
										if(il + iil < NL){
											double nu0 = nuP[il + iil] - cut;
											double nu1 = nuP[il + iil] + cut;

											nu11 = fmax(nu11, nu1);
											nu00 = fmin(nu00, nu0);
										}
									}
									for(int bin = 0; bin < param.nbins; ++bin){
										if(binBoundaries_h[bin + 1] > nu11){
											double dnu = (binBoundaries_h[bin + 1] - binBoundaries_h[bin]) / ((double)(param.Nxb));
											int bstart = bin * param.Nxb;
											ii11 = (nu11 - binBoundaries_h[bin]) / dnu + bstart + 2;
											break;
										}
									}
									for(int bin = 0; bin < param.nbins; ++bin){
										if(binBoundaries_h[bin + 1] > nu00){
											double dnu = (binBoundaries_h[bin + 1] - binBoundaries_h[bin]) / ((double)(param.Nxb));
											int bstart = bin * param.Nxb;
											ii00 = (nu00 - binBoundaries_h[bin]) / dnu + bstart - 1;
											break;
										}
									}
								}

								ii11 = min(Nx, ii11);
								ii00 = max(0, ii00);

								int nt = ii11 - ii00;
								int nstart = ii00;
								int nll = min(nl, NL - il);	
	if(il % 10000 == 0) printf("A %d %d %d %d %d\n",il, ii00, ii11, nll, nt);
								for(int k = 0; k < nt; k += def_nthmax){
									int Nk = min(def_nthmax, nt - k);
									if(Nk > 0 && nll > 0){
										Line2f_kernel < nl, 0 > <<< (max(Nk, nll) + nl - 1) / nl, nl >>> (L.S1f_d, L.vyf_d, L.va_d, L.vb_d, L.vcut2_d, K_d + iP * Nx, il, nstart, Nk, nll, param.useIndividualX, param.Nxb, binBoundaries_d, 0.0f, 0.0f, 0.0f, param.profile);
									}
									nstart += def_nthmax;
								}
							}
						} //end  param.RLOW == 0
						else{
							//lower resolution
							const int nl = 512;
							for(int il = 0; il < NL; il += nl){ //loop over lines
								int ii11 = 0;
								int ii00 = Nx1;
								for(int iil = 0; iil < nl; ++iil){
									if(il + iil < NL){
										int Inu = (int)((nuP[il + iil] - param.numin) / (param.dnu * 10));
										int ii0 = Inu - (int)(cut / (param.dnu * 10)) - 1;
										int ii1 = Inu + (int)(cut / (param.dnu * 10)) + 2;
	//if(iil % 10000 == 0) printf("%d %.30g %d %d %d\n", il + iil, nuP[il + iil], Inu, ii0, ii1);

										ii11 = max(ii11, ii1);
										ii00 = min(ii00, ii0);
									}
								}
								ii11 = min(Nx1, ii11);
								ii00 = max(0, ii00);
								int nt = ii11 - ii00;
								int nstart = ii00;
								int nll = min(nl, NL - il);	
	if(il % 10000 == 0) printf("Ac %d %d %d %d %d\n",il, ii00, ii11, nll, nt);
								for(int k = 0; k < nt; k += def_nthmax){
									int Nk = min(def_nthmax, nt - k);
									if(Nk > 0 && nll > 0){
										Line2f_kernel < nl, -1 > <<< (max(Nk, nll) + nl - 1) / nl, nl >>> (L.S1f_d, L.vyf_d, L.va_d, L.vb_d, L.vcut2_d, K1_d, il, nstart, Nk, nll, param.useIndividualX, (param.Nxb + 9) / 10, binBoundaries_d, 0.0f, 0.0f, 0.0f, param.profile);
									}
									nstart += def_nthmax;
								}
							}
							//lower resolution interpolation correction
							const int nlb = 512;
							for(int il = 0; il < NL; il += nlb){ //loop over lines
								int ii11 = 0;
								int ii00 = Nx;
								for(int iil = 0; iil < nlb; ++iil){
									if(il + iil < NL){

										double aD2 = 1.0 / (ialphaDP[il + iil] * ialphaDP[il + iil]);
										double aL2 = vyP[il + iil] * vyP[il + iil] * aD2;
										double Dnu2 = 1.0e6 * aD2 - aL2;
										double Dnu = 0.0;
										if(Dnu2 > 0.0){
											Dnu = sqrt(Dnu2);
											int Inu = (int)((nuP[il + iil] - param.numin) / (param.dnu));
											int ii0 = ((Inu - (int)(Dnu / (param.dnu))) / 10) * 10;
											int ii1 = ii0 + 12;

											ii11 = max(ii11, ii1);
											ii00 = min(ii00, ii0);
										}
									}
								}
								ii11 = min(Nx, ii11);
								ii00 = max(0, ii00);
								int nt = ii11 - ii00;
								int nstart = ii00;
								int nll = min(nlb, NL - il);	
	if(il % 10000 == 0) printf("Bcl %d %d %d %d %d\n",il, ii00, ii11, nll, nt);
								for(int k = 0; k < nt; k += def_nthmax){
									int Nk = min(def_nthmax, nt - k);
									if(Nk > 0 && nll > 0){
										Line2f_kernel < nlb, 10 > <<< (max(Nk, nll) + nlb - 1) / nlb, nlb >>> (L.S1f_d, L.vyf_d, L.va_d, L.vb_d, L.vcut2_d, Kc_d, il, nstart, Nk, nll, param.useIndividualX, param.Nxb, binBoundaries_d, 0.0f, 0.0f, 0.0f, param.profile);
									}
									nstart += def_nthmax;
								}
							}
							for(int il = 0; il < NL; il += nlb){ //loop over lines
								int ii11 = 0;
								int ii00 = Nx;
								for(int iil = 0; iil < nlb; ++iil){
									if(il + iil < NL){

										double aD2 = 1.0 / (ialphaDP[il + iil] * ialphaDP[il + iil]);
										double aL2 = vyP[il + iil] * vyP[il + iil] * aD2;
										double Dnu2 = 1.0e6 * aD2 - aL2;
										double Dnu = 0.0;
										if(Dnu2 > 0.0){
											Dnu = sqrt(Dnu2);
											int Inu = (int)((nuP[il + iil] - param.numin) / (param.dnu));
											int ii0 = ((Inu + (int)(Dnu / (param.dnu))) / 10) * 10;
											int ii1 = ii0 + 12;

											ii11 = max(ii11, ii1);
											ii00 = min(ii00, ii0);
										}
									}
								}
								ii11 = min(Nx, ii11);
								ii00 = max(0, ii00);
								int nt = ii11 - ii00;
								int nstart = ii00;
								int nll = min(nlb, NL - il);	
	if(il % 10000 == 0) printf("Bcr %d %d %d %d %d\n",il, ii00, ii11, nll, nt);
								for(int k = 0; k < nt; k += def_nthmax){
									int Nk = min(def_nthmax, nt - k);
									if(Nk > 0 && nll > 0){
										Line2f_kernel < nlb, 11 > <<< (max(Nk, nll) + nlb - 1) / nlb, nlb >>> (L.S1f_d, L.vyf_d, L.va_d, L.vb_d, L.vcut2_d, Kc_d, il, nstart, Nk, nll, param.useIndividualX, param.Nxb, binBoundaries_d, 0.0f, 0.0f, 0.0f, param.profile);
									}
									nstart += def_nthmax;
								}
							}
							for(int il = 0; il < NL; il += nlb){ //loop over lines
								int ii11 = 0;
								int ii00 = Nx;
								for(int iil = 0; iil < nlb; ++iil){
									if(il + iil < NL){
										int Inu = (int)((nuP[il + iil] - param.numin) / param.dnu);
										int ii0 = (Inu + (int)(cut / param.dnu)) / 10 * 10;
										int ii1 = ii0 + 12;
	//if(iil % 10000 == 0) printf("%d %.30g %d %d %d\n", il + iil, nuP[il + iil], Inu, ii0, ii1);

										ii11 = max(ii11, ii1);
										ii00 = min(ii00, ii0);
									}
								}
								ii11 = min(Nx, ii11);
								ii00 = max(0, ii00);
								int nt = ii11 - ii00;
								int nstart = ii00;
								int nll = min(nlb, NL - il);	
	if(il % 10000 == 0) printf("Acr %d %d %d %d %d\n",il, ii00, ii11, nll, nt);
								for(int k = 0; k < nt; k += def_nthmax){
									int Nk = min(def_nthmax, nt - k);
									if(Nk > 0 && nll > 0){
										Line2f_kernel < nlb, 12 > <<< (max(Nk, nll) + nlb - 1) / nlb, nlb >>> (L.S1f_d, L.vyf_d, L.va_d, L.vb_d, L.vcut2_d, Kc_d, il, nstart, Nk, nll, param.useIndividualX, param.Nxb, binBoundaries_d, 0.0f, 0.0f, 0.0f, param.profile);
									}
									nstart += def_nthmax;
								}
							}
							for(int il = 0; il < NL; il += nlb){ //loop over lines
								int ii11 = 0;
								int ii00 = Nx;
								for(int iil = 0; iil < nlb; ++iil){
									if(il + iil < NL){
										int Inu = (int)((nuP[il + iil] - param.numin) / param.dnu);
										int ii0 = (Inu - (int)(cut / param.dnu)) / 10 * 10;
										int ii1 = ii0 + 12;
	//if(iil % 10000 == 0) printf("%d %.30g %d %d %d\n", il + iil, nuP[il + iil], Inu, ii0, ii1);

										ii11 = max(ii11, ii1);
										ii00 = min(ii00, ii0);
									}
								}
								ii11 = min(Nx, ii11);
								ii00 = max(0, ii00);
								int nt = ii11 - ii00;
								int nstart = ii00;
								int nll = min(nlb, NL - il);	
	if(il % 10000 == 0) printf("Acl %d %d %d %d %d\n",il, ii00, ii11, nll, nt);
								for(int k = 0; k < nt; k += def_nthmax){
									int Nk = min(def_nthmax, nt - k);
									if(Nk > 0 && nll > 0){
										Line2f_kernel < nlb, 13 > <<< (max(Nk, nll) + nlb - 1) / nlb, nlb >>> (L.S1f_d, L.vyf_d, L.va_d, L.vb_d, L.vcut2_d, Kc_d, il, nstart, Nk, nll, param.useIndividualX, param.Nxb, binBoundaries_d, 0.0f, 0.0f, 0.0f, param.profile);
									}
									nstart += def_nthmax;
								}
							}
							for(int k = 0; k < Nx; k += def_nthmax){
								int Nk = min(def_nthmax, Nx - k);
								InterpolateX2_kernel <<< (Nk + 511) / 512, 512 >>> (K_d + iP * Nx, Kc_d, Nx, param.Nxb, param.useIndividualX, binBoundaries_d, k);
								InterpolateX1_kernel <<< (Nk + 511) / 512, 512 >>> (K_d + iP * Nx, K1_d, Nx, param.Nxb, param.useIndividualX, binBoundaries_d, k);
							}
						} // end param.RLOW == 1
						//search second order regimes of the Voigt profile
						const int nl2 = 512;
						for(int il = 0; il < NL; il += nl2){ //loop over lines
							int ii11 = 0;
							int ii00 = Nx;
							if(param.useIndividualX == 0){
								for(int iil = 0; iil < nl2; ++iil){
									if(il + iil < NL){

										double aD2 = 1.0 / (ialphaDP[il + iil] * ialphaDP[il + iil]);
										double aL2 = vyP[il + iil] * vyP[il + iil] * aD2;
										double Dnu2 = 1.0e6 * aD2 - aL2;
										double Dnu = 0.0;
										if(Dnu2 > 0.0){
											Dnu = sqrt(Dnu2);
											int Inu = (int)((nuP[il + iil] - param.numin) / param.dnu);
											int ii0 = Inu - (int)(Dnu / param.dnu) - 1;
											int ii1 = Inu + (int)(Dnu / param.dnu) + 2;
	//printf("%d %d %d\n", il + iil, ii0, ii1);

											ii11 = max(ii11, ii1);
											ii00 = min(ii00, ii0);
										}
									}
								}
							}
							else{
								double nu00 = param.numax;
								double nu11 = param.numin;
								for(int iil = 0; iil < nl2; ++iil){
									if(il + iil < NL){
										double aD2 = 1.0 / (ialphaDP[il + iil] * ialphaDP[il + iil]);
										double aL2 = vyP[il + iil] * vyP[il + iil] * aD2;
										double Dnu2 = 1.0e6 * aD2 - aL2;
										double Dnu = 0.0;
										if(Dnu2 > 0.0){
											Dnu = sqrt(Dnu2);
											double nu0 = nuP[il + iil] - Dnu;
											double nu1 = nuP[il + iil] + Dnu;

											nu11 = fmax(nu11, nu1);
											nu00 = fmin(nu00, nu0);
										}
									}
								}
								for(int bin = 0; bin < param.nbins; ++bin){
									if(binBoundaries_h[bin + 1] > nu11){
										double dnu = (binBoundaries_h[bin + 1] - binBoundaries_h[bin]) / ((double)(param.Nxb));
										int bstart = bin * param.Nxb;
										ii11 = (nu11 - binBoundaries_h[bin]) / dnu + bstart + 2;
										break;
									}
								}
								for(int bin = 0; bin < param.nbins; ++bin){
									if(binBoundaries_h[bin + 1] > nu00){
										double dnu = (binBoundaries_h[bin + 1] - binBoundaries_h[bin]) / ((double)(param.Nxb));
										int bstart = bin * param.Nxb;
										ii00 = (nu00 - binBoundaries_h[bin]) / dnu + bstart - 1;
										break;
									}
								}
							}

							ii11 = min(Nx, ii11);
							ii00 = max(0, ii00);
							int nt = ii11 - ii00;
							int nstart = ii00;
							int nll = min(nl2, NL - il);	
if(il % 10000 == 0) printf("B %d %d %d %d %d\n",il, ii00, ii11, nll, nt);
							for(int k = 0; k < nt; k += def_nthmax){
								int Nk = min(def_nthmax, nt - k);
								if(Nk > 0 && nll > 0){
									Line2f_kernel < nl2, 1 > <<< (max(Nk, nll) + nl2 - 1) / nl2, nl2 >>> (L.S1f_d, L.vyf_d, L.va_d, L.vb_d, L.vcut2_d, K_d + iP * Nx, il, nstart, Nk, nll, param.useIndividualX, param.Nxb, binBoundaries_d, 0.0f, 0.0f, 0.0f, param.profile);
								}
								nstart += def_nthmax;
							}
						}

						//search higher order regimes of the Voigt profile
						const int nl3 = 512;
						float a = (float)(M_PI * sqrt(-1.0 / log(def_TOLF * 0.5)));
						float b = (float)(1.0 / sqrt(M_PI));
						float c = (float)(2.0 * a / M_PI);

						for(int il = 0; il < NL; il += nl3){ //loop over lines
							int ii11 = 0;
							int ii00 = Nx;
							if(param.useIndividualX == 0){
								for(int iil = 0; iil < nl3; ++iil){
									if(il + iil < NL){

										double aD2 = 1.0 / (ialphaDP[il + iil] * ialphaDP[il + iil]);
										double aL2 = vyP[il + iil] * vyP[il + iil] * aD2;
										double Dnu2 = 1.0e2 * aD2 - aL2;
										double Dnu = 0.0;
										if(Dnu2 > 0.0){
											Dnu = sqrt(Dnu2);
											int Inu = (int)((nuP[il + iil] - param.numin) / param.dnu);
											int ii0 = Inu - (int)(Dnu / param.dnu) - 1;
											int ii1 = Inu + (int)(Dnu / param.dnu) + 2;
	//printf("%d %d %d\n", il + iil, ii0, ii1);
											ii11 = max(ii11, ii1);
											ii00 = min(ii00, ii0);
										}
									}
								}
							}
							else{
								double nu00 = param.numax;
								double nu11 = param.numin;
								for(int iil = 0; iil < nl3; ++iil){
									if(il + iil < NL){
										double aD2 = 1.0 / (ialphaDP[il + iil] * ialphaDP[il + iil]);
										double aL2 = vyP[il + iil] * vyP[il + iil] * aD2;
										double Dnu2 = 1.0e2 * aD2 - aL2;
										double Dnu = 0.0;
										if(Dnu2 > 0.0){
											Dnu = sqrt(Dnu2);
											double nu0 = nuP[il + iil] - Dnu;
											double nu1 = nuP[il + iil] + Dnu;

											nu11 = fmax(nu11, nu1);
											nu00 = fmin(nu00, nu0);
										}
									}
								}
								for(int bin = 0; bin < param.nbins; ++bin){
									if(binBoundaries_h[bin + 1] > nu11){
										double dnu = (binBoundaries_h[bin + 1] - binBoundaries_h[bin]) / ((double)(param.Nxb));
										int bstart = bin * param.Nxb;
										ii11 = (nu11 - binBoundaries_h[bin]) / dnu + bstart + 2;
										break;
									}
								}
								for(int bin = 0; bin < param.nbins; ++bin){
									if(binBoundaries_h[bin + 1] > nu00){
										double dnu = (binBoundaries_h[bin + 1] - binBoundaries_h[bin]) / ((double)(param.Nxb));
										int bstart = bin * param.Nxb;
										ii00 = (nu00 - binBoundaries_h[bin]) / dnu + bstart - 1;
										break;
									}
								}
							}
							ii11 = min(Nx, ii11);
							ii00 = max(0, ii00);
							int nt = ii11 - ii00;
							int nstart = ii00;
							int nll = min(nl3, NL - il);	
if(il % 10000 == 0) printf("C %d %d %d %d %d\n",il, ii00, ii11, nll, nt);
							for(int k = 0; k < nt; k += def_nthmax){
								int Nk = min(def_nthmax, nt - k);
								if(Nk > 0 && nll > 0){
									Line2f_kernel < nl3, 2 > <<< (max(Nk, nll) + nl3 - 1) / nl3, nl3 >>> (L.Sf_d, L.vyf_d, L.va_d, L.vb_d, L.vcut2_d, K_d + iP * Nx, il, nstart, Nk, nll, param.useIndividualX, param.Nxb, binBoundaries_d, a, b, c, param.profile);
								}
								nstart += def_nthmax;
							}
						}

					} //end Nx > def_NXLOW
					//*************************************
					hipEventRecord(stop);
					if(iP < param.nP - 1){
						//synchronize here only if no more data has to be read from the disk.
						//otherwise read data before synchronization
						hipEventSynchronize(stop);
						hipEventElapsedTime(&milliseconds, start, stop);

						time[2] += milliseconds * 0.001;
						if(iP == param.nP - 1){
							printf("Time for K(x):         %g seconds\n", time[2]);
						}
			
						hipDeviceSynchronize();
						error = hipGetLastError();
						if(error != 0){
							printf("Kb error = %d = %s\n",error, hipGetErrorString(error));
							return 0;
						}
						gettimeofday(&tt1, NULL);
					}

				} // End of pressure loop

			} // End of maxLines loop
		} // End of linefile loop

		hipEventSynchronize(stop);
		if(m.NL[fi] > 0){
			hipEventElapsedTime(&milliseconds, start, stop);

			time[2] += milliseconds * 0.001;
		}
		printf("Time for K(x):         %g seconds\n", time[2]);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != 0){
			printf("Kc error = %d = %s\n",error, hipGetErrorString(error));
			return 0;
		}
			InfoFile = fopen(InfoFilename, "a");
			fprintf(InfoFile,"File %d of %d\n", fi, fi);
			fprintf(InfoFile,"Number of lines: %lld\n", m.NL[fi]);
			fprintf(InfoFile,"Time for input:        %g seconds\n", time[0]);
			fprintf(InfoFile,"Time for Lines:        %g seconds\n", time[1]);
			fprintf(InfoFile,"Time for K(x):         %g seconds\n", time[2]);
			fclose(InfoFile);
		gettimeofday(&tt1, NULL);

		free(nuP);
		free(ialphaDP);
		free(vyP);
	}
	hipFree(Limits_d);
	hipFree(MaxLimits_d);
	free(binBoundaries_h);
	hipFree(binIndex_d);
	hipFree(binBoundaries_d);	
	hipFree(K1_d);
	hipFree(Kc_d);

	//***************************
	//Write the full line profile
	//****************************
	if(param.doStoreFullK == 1){
		FILE *OutFile;
		char OutFilename[160];
		sprintf(OutFilename, "Out_%s.dat", param.name);
			
		OutFile = fopen(OutFilename, filemode);

		for(int iP = 0; iP < param.nP; ++iP){
			hipMemcpy(K_h, K_d + iP * Nx, Nx * sizeof(double), hipMemcpyDeviceToHost);
			for(int j = 0; j < Nx; ++j){

				if(param.nP == 1){
					fprintf(OutFile, "%.20g %.20g\n", x_h[j], K_h[j] * unitScale);
				}
				else{
					fprintf(OutFile, "%.20g %.20g %.20g %.20g\n", x_h[j], K_h[j] * unitScale, param.T, P_h[iP]);
				}
			}
			fprintf(OutFile, "\n\n");
		}
		fclose(OutFile);
	}
	if(param.doStoreFullK == -1){
		FILE *OutFile;
		char OutFilename[160];
		sprintf(OutFilename, "%sOut_%s.dat", param.pathK, param.name);
			
		OutFile = fopen(OutFilename, "r");
		if(OutFile == NULL){
			printf("Error: Input file not found %s\n", OutFilename);
			return 0;
		}

		for(int iP = 0; iP < param.nP; ++iP){
			for(int j = 0; j < Nx; ++j){

				if(param.nP == 1){
					double k;
					fscanf(OutFile, "%lf %lf\n", &x_h[j], &k);
					K_h[j] = k / unitScale;
				}
				else{
					double k, t, p;
					fscanf(OutFile, "%lf %lf %lf %lf\n", &x_h[j], &k, &t, &p);
					K_h[j] = k / unitScale;
				}
			}
			hipMemcpy(K_d + iP * Nx, K_h, Nx * sizeof(double), hipMemcpyHostToDevice);
			fscanf(OutFile, "\n\n");
		}
		fclose(OutFile);
	}
	if(param.doStoreFullK == 2){
		//write a binary file in single precision
		FILE *OutFile;
		char OutFilename[160];
		sprintf(OutFilename, "Out_%s.bin", param.name);
			
		if(param.replaceFiles == 0){
			OutFile = fopen(OutFilename, "ab");
		}
		else{
			OutFile = fopen(OutFilename, "wb");
		}

		for(int iP = 0; iP < param.nP; ++iP){
			hipMemcpy(K_h, K_d + iP * Nx, Nx * sizeof(double), hipMemcpyDeviceToHost);
			for(int j = 0; j < Nx; ++j){
				float Kf = (float)(K_h[j]);
				fwrite(&Kf, sizeof(float), 1, OutFile);
			}
		}
		fclose(OutFile);
	}
	if(param.doStoreFullK == -2){
		//read a binary file
		FILE *OutFile;
		char OutFilename[160];
		sprintf(OutFilename, "%sOut_%s.bin", param.pathK, param.name);
			
		OutFile = fopen(OutFilename, "rb");
		if(OutFile == NULL){
			printf("Error: Input file not found %s\n", OutFilename);
			return 0;
		}

		for(int iP = 0; iP < param.nP; ++iP){
			for(int j = 0; j < Nx; ++j){
				float Kf;
				fread(&Kf, sizeof(float), 1, OutFile);
				K_h[j] = (double)(Kf);
			}
			hipMemcpy(K_d + iP * Nx, K_h, Nx * sizeof(double), hipMemcpyHostToDevice);
		}
		fclose(OutFile);
	}
	//*******************************

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != 0){
		printf("Write error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}
	gettimeofday(&tt2, NULL);
	times = (tt2.tv_sec - tt1.tv_sec);
	timems = (tt2.tv_usec - tt1.tv_usec);

	time[3] = times + timems/1000000.0;
	printf("Time for write K(x):   %g seconds\n", time[3]);

	gettimeofday(&tt1, NULL);

	//**************************************
	//compute the Planck and Rosseland means
	//**************************************
	if(param.doMean > 0){
		
		double *Pmn_d;
		double *Rmn_d;

		hipMalloc((void **) &Pmn_d, Nx * sizeof(double));
		hipMalloc((void **) &Rmn_d, Nx * sizeof(double));
	
		double *means_h, *means_d;	
		means_h = (double*)malloc(4 * sizeof(double));
		hipMalloc((void **) &means_d, 4 * sizeof(double));

		FILE *Out4File;
		char Out4Filename[160];

		sprintf(Out4Filename, "Out_%s_mean.dat", param.name);
		Out4File = fopen(Out4Filename, filemode);
	
		for(int iP = 0; iP < param.nP; ++iP){

			Mean_kernel <<< (Nx + 511) / 512, 512 >>> (x_d, Pmn_d, Rmn_d, param.T, Nx);
/*
printf("\n\n");
hipMemcpy(K_h, Pmn_d, Nx * sizeof(double), hipMemcpyDeviceToHost);
for(int i = 0; i < Nx; ++i){
	printf("%g %g\n", param.numin + i * param.dnu, K_h[i]);
}
printf("\n\n");
hipMemcpy(K_h, Rmn_d, Nx * sizeof(double), hipMemcpyDeviceToHost);
for(int i = 0; i < Nx; ++i){
	printf("%g %g\n", param.numin + i * param.dnu, K_h[i]);
}
printf("\n\n");
*/
			IntegrateMean_kernel <512> <<< 4, 512 >>> (Pmn_d, Rmn_d, x_d, K_d + iP * Nx, means_d, Nx, param.useIndividualX);
			double sigma = 2.0 * def_kB * def_kB * def_kB * def_kB / ( def_h * def_h * def_h * def_c * def_c * 15.0) * M_PI * M_PI * M_PI * M_PI * M_PI;
			double integral1 = sigma * param.T * param.T * param.T * param.T / M_PI;
			double integral2 = M_PI / (4.0 * sigma * param.T * param.T * param.T);
		
			hipMemcpy(means_h, means_d, 4 * sizeof(double), hipMemcpyDeviceToHost);


			if(param.nP == 1){
				fprintf(Out4File, "%.20g\n", means_h[0] / means_h[2]);
				fprintf(Out4File, "%.20g\n", means_h[3] / means_h[1]);
				fprintf(Out4File, "%.20g\n", means_h[2]);
				fprintf(Out4File, "%.20g\n", integral1);
				fprintf(Out4File, "%.20g\n", means_h[3]);
				fprintf(Out4File, "%.20g\n", 1.0 / integral2);
			}
			else{
				fprintf(Out4File, "%.20g %.20g %.20g\n", means_h[0] / means_h[2], param.T, P_h[iP]);
				fprintf(Out4File, "%.20g %.20g %.20g\n", means_h[3] / means_h[1], param.T, P_h[iP]);
				fprintf(Out4File, "%.20g %.20g %.20g\n", means_h[2], param.T, P_h[iP]);
				fprintf(Out4File, "%.20g %.20g %.20g\n", integral1, param.T, P_h[iP]);
				fprintf(Out4File, "%.20g %.20g %.20g\n", means_h[3], param.T, P_h[iP]);
				fprintf(Out4File, "%.20g %.20g %.20g\n", 1.0 / integral2, param.T, P_h[iP]);

			}
			//fprintf(Out4File, "\n\n");
		}
		
		fclose(Out4File);

		free(means_h);
		hipFree(means_d);
		hipFree(Pmn_d);
		hipFree(Rmn_d);
	}
	hipFree(x_d);
	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != 0){
		printf("maen error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}
	gettimeofday(&tt2, NULL);
	times = (tt2.tv_sec - tt1.tv_sec);
	timems = (tt2.tv_usec - tt1.tv_usec);

	time[4] = times + timems/1000000.0;
	printf("Time for mean K(x):    %g seconds\n", time[4]);

	gettimeofday(&tt1, NULL);



	//***************************************
	//Do the sorting of K for all bins
	//***************************************
	thrust::device_ptr<double> K_dt = thrust::device_pointer_cast(K_d);
	thrust::device_ptr<int> binKey_dt = thrust::device_pointer_cast(binKey_d);
	for(int iP = 0; iP < param.nP; ++iP){
		thrust::sort_by_key(K_dt + iP * Nx, K_dt + Nx + iP * Nx, binKey_dt);
		thrust::stable_sort_by_key(binKey_dt, binKey_dt + Nx, K_dt + iP * Nx);
	}
	hipFree(binKey_d);
	//****************************************

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != 0){
		printf("Sort error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}
	gettimeofday(&tt2, NULL);
	times = (tt2.tv_sec - tt1.tv_sec);
	timems = (tt2.tv_usec - tt1.tv_usec);

	time[5] = times + timems/1000000.0;
	printf("Time for sort K(x):    %g seconds\n", time[5]);

	gettimeofday(&tt1, NULL);

	//*********************************
	//Prepare Resampling and do QR factorization, the same for all bins
	// this doesn't work with individual bins
	//*********************************

//size_t free_byte;
//size_t total_byte;
//hipMemGetInfo( &free_byte, &total_byte );
//printf("***MEMRORY %g %g %g\n", (double)(free_byte), (double)(total_byte), (double)(total_byte) - (double)(free_byte));
	int *Nxmin_h, *Nxmin_d;		
	Nxmin_h = (int*)malloc(param.nbins * sizeof(int));
	hipMalloc((void **) &Nxmin_d, param.nbins * sizeof(int));
	for(int i = 0; i < param.nbins; ++i){
		Nxmin_h[i] = 0;
	}
	hipMemset(Nxmin_d, 0, param.nbins * sizeof(int));
	if(param.doResampling > 0){

		double *K2_h, *K2_d;
		K2_h = (double*)malloc(Nx * sizeof(double));
		hipMalloc((void **) &K2_d, Nx * sizeof(double));
//hipMemGetInfo( &free_byte, &total_byte );
//printf("***MEMRORY %g %g %g\n", (double)(free_byte), (double)(total_byte), (double)(total_byte) - (double)(free_byte));

		double *V_d;			//Vandermonde like matrix for least sqaures
		double *C_d, *D_d;

		hipMalloc((void **) &V_d, param.nC * param.Nxb * sizeof(double));
		hipMalloc((void **) &C_d, param.nC * sizeof(double));
		hipMalloc((void **) &D_d, param.nC * sizeof(double));

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != 0){
			printf("Resampling Allocation error = %d = %s\n",error, hipGetErrorString(error));
			return 0;
		}

		Vandermonde_kernel <<< (param.Nxb + 511) / 512, 512 >>> (V_d, (double)(param.Nxb), param.nC);
		QR_kernel <512> <<< 1, 512 >>> (V_d, C_d, D_d, param.Nxb, param.nC);

		FILE *Out3File;
		char Out3Filename[160];
		if(param.doResampling == 1){
			sprintf(Out3Filename, "Out_%s_cbin.dat", param.name);
			Out3File = fopen(Out3Filename, filemode);
		}
		if(param.doResampling == 2){
			if(param.replaceFiles == 1){
				for(int i = 0; i < param.nbins; ++i){
					sprintf(Out3Filename, "Out_%s_cbin%.4d.dat", param.name, i);
					Out3File = fopen(Out3Filename, "w");
					fclose(Out3File);	
				}
			}
			sprintf(Out3Filename, "Out_%s_cbin%.4d.dat", param.name, 0);
			Out3File = fopen(Out3Filename, "a");
		}

		for(int iP = 0; iP < param.nP; ++iP){	
			if(param.doResampling == 2 && iP > 0){
				fclose(Out3File);
				sprintf(Out3Filename, "Out_%s_cbin%.4d.dat", param.name, 0);
				Out3File = fopen(Out3Filename, "a");
			}
			hipMemset(K2_d, 0, Nx * sizeof(double));
			hipMemset(Nxmin_d, 0, param.nbins * sizeof(int));

			findCut_kernel <<< (Nx + 511) / 512, 512 >>> (K_d + iP * Nx, Nx, param.Nxb, param.kmin, Nxmin_d, param.nbins);
			rescale_kernel < 512 > <<< param.nbins, 512 >>> (Nxmin_d, K_d + iP * Nx, K2_d, param.Nxb, param.kmin, 1);
/*
hipMemcpy(K2_h, K2_d, Nx * sizeof(double), hipMemcpyDeviceToHost);
hipMemcpy(K_h, K_d + iP * Nx, Nx * sizeof(double), hipMemcpyDeviceToHost);
hipDeviceSynchronize();
//printf only cut and empty bins
for(int i = 0; i < param.nbins; ++i){
	int il = i * param.Nxb;
	if(K_h[il] == param.kmin){
		for(int j = 0; j < param.Nxb; ++j){
//			printf("%g %.20g\n", j / (double)(param.Nxb), K2_h[j + il]);
		}
//		printf("\n\n");
	}
}
//print all bins
for(int i = 0; i < Nx; ++i){
	printf("%d %.20g %.20g\n", i, K_h[i], K2_h[i]);
}
*/
			copyK2_kernel< 512 > <<< param.nbins, 512 >>> (Nxmin_d, K_d + iP * Nx, K2_d, param.Nxb);
			hipMemcpy(Nxmin_h, Nxmin_d, param.nbins * sizeof(int), hipMemcpyDeviceToHost);
	
			lnK_kernel <<< (Nx + 511) / 512, 512 >>> (K_d + iP * Nx, Nx);
			leastSquare_kernel <512> <<< param.nbins, 512 >>> (V_d, C_d, D_d, K_d + iP * Nx, param.Nxb, param.nC);

			for(int i = 0; i < param.nbins; ++i){
				int il = i * param.Nxb;
				hipMemcpy(K_h + il, K_d + il + iP * Nx, param.nC * sizeof(double), hipMemcpyDeviceToHost);
		
				fprintf(Out3File, "%.20g %.20g ", param.kmin, fmin(Nxmin_h[i] / ((double)(param.Nxb - 1)), 1.0));
				for(int ic = 0; ic < param.nC; ++ic){
					if(Nxmin_h[i] != param.Nxb) fprintf(Out3File, "%.20g ", K_h[il + ic]);
					else fprintf(Out3File, "0.0 ");
				}
				if(param.nP > 1){
					fprintf(Out3File, "%.20g %.20g ", param.T, P_h[iP]);
				}
				if(param.doResampling == 1){
					fprintf(Out3File, "\n\n");
				}
				if(param.doResampling == 2 && i < param.nbins - 1){
					fprintf(Out3File, "\n");
					fclose(Out3File);
					sprintf(Out3Filename, "Out_%s_cbin%.4d.dat", param.name, i + 1);
					Out3File = fopen(Out3Filename, "a");
				}
			}
			//fprintf(Out3File, "\n\n");
			if(param.doTransmission > 0 || param.doStoreK > 0){
				expfx_kernel <<< param.nbins, 512 >>> (K_d + iP * Nx, param.nC, param.Nxb);
				rescale_kernel < 512 > <<< param.nbins, 512 >>> (Nxmin_d, K_d + iP * Nx, K2_d, param.Nxb, param.kmin, -1);
				copyK2_kernel< 512 > <<< param.nbins, 512 >>> (Nxmin_d, K_d + iP * Nx, K2_d, param.Nxb);
			}	
		}
		fclose(Out3File);
		hipFree(V_d);
		hipFree(C_d);
		hipFree(D_d);
		hipFree(K2_d);
		free(K2_h);
	}
	//**********************************
	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != 0){
		printf("Resampling error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}
	gettimeofday(&tt2, NULL);
	times = (tt2.tv_sec - tt1.tv_sec);
	timems = (tt2.tv_usec - tt1.tv_usec);

	time[6] = times + timems/1000000.0;
	printf("Time for Resampling:   %g seconds\n", time[6]);

	gettimeofday(&tt1, NULL);

	//*****************************
	//Write K per bin output
	//*****************************
	if(param.doStoreK > 0){
		FILE *Out2File;
		char Out2Filename[160];
		if(param.doStoreK == 1){
			sprintf(Out2Filename, "Out_%s_bin.dat", param.name);
			Out2File = fopen(Out2Filename, filemode);
		}
		if(param.doStoreK == 2){
			if(param.replaceFiles == 1){
				for(int i = 0; i < param.nbins; ++i){
					sprintf(Out2Filename, "Out_%s_bin%.4d.dat", param.name, i);
					Out2File = fopen(Out2Filename, "w");
					fclose(Out2File);	
				}
			}
			sprintf(Out2Filename, "Out_%s_bin%.4d.dat", param.name, 0);
			Out2File = fopen(Out2Filename, "a");
		}
		
		for(int iP = 0; iP < param.nP; ++iP){
			if(param.doStoreK == 2 && iP > 0){
				fclose(Out2File);
				sprintf(Out2Filename, "Out_%s_bin%.4d.dat", param.name, 0);
				Out2File = fopen(Out2Filename, "a");
			}
			hipMemcpy(K_h, K_d + iP * Nx, Nx * sizeof(double), hipMemcpyDeviceToHost);
			if(param.useIndividualBins == 0){
				for(int i = 0; i < param.nbins; ++i){
					int Nxb = param.Nxb;
					int il = i * Nxb;
					int iedge = 0; //index of edge
					int nedge = 0; //number of points per edge intervall
					double sedge = 0.0; //sum of points in edge intervall
					for(int j = 0; j < Nxb; ++j){
						double y = j / ((double)(Nxb - 1));
						double y1 = (j + 1) / ((double)(Nxb - 1));
						if(param.useOutputEdges == 0){
							if(param.nP == 1){
								fprintf(Out2File, "%g %.20g\n", y, K_h[j + il] * unitScale);
							}
							else{
								fprintf(Out2File, "%g %.20g %g %g %d\n", y, K_h[j + il] * unitScale, param.T, P_h[iP], j);
							}
						}
						else{
							double edge = outputEdges_h[iedge];
							++nedge;
							sedge += K_h[j + il] * unitScale;
							if(y <= edge && edge <= y1 && iedge < param.nedges){
								if(param.nP == 1){
									if(iedge > 0) fprintf(Out2File, "%g %.20g\n", 0.5 * (edge + outputEdges_h[iedge - 1]), sedge / ((double)(nedge)));
								}
								else{
									if(iedge > 0) fprintf(Out2File, "%g %.20g %g %g %d\n", 0.5 * (edge + outputEdges_h[iedge - 1]), sedge / ((double)(nedge)), param.T, P_h[iP], iedge - 1);

								}
								++iedge;
								nedge = 0;
								sedge = 0.0;
							}
						}
					}
					if(param.doStoreK == 1){
						fprintf(Out2File,"\n\n");
					}
					if(param.doStoreK == 2 && i < param.nbins - 1){
						fclose(Out2File);
						sprintf(Out2Filename, "Out_%s_bin%.4d.dat", param.name, i + 1);
						Out2File = fopen(Out2Filename, "a");
					}
				}
			}
			else{
				int ib = 0;
				int j = 0;
				int iedge = 0; //inde of edge
				int nedge = 0; //number of points per edge intervall
				double sedge = 0.0; //sum of points in edge intervall
				for(int i = 0; i < Nx; ++i){
					int il = binIndex_h[ib];
					int ir = binIndex_h[ib + 1];
					int Nxb = ir - il;

					double y = j / ((double)(Nxb - 1));
					double y1 = (j + 1) / ((double)(Nxb - 1));

					if(param.useOutputEdges == 0){
						if(param.nP == 1){
							fprintf(Out2File, "%g %.20g\n", y, K_h[i] * unitScale);
						}
						else{
							fprintf(Out2File, "%g %.20g %.20g %.20g %d\n", y, K_h[i] * unitScale, param.T, P_h[iP], j);
						}
					}
					else{
						double edge = outputEdges_h[iedge];
						++nedge;
						sedge += K_h[i] * unitScale;
						if(y <= edge && edge <= y1 && iedge < param.nedges){
							if(param.nP == 1){
								if(iedge > 0) fprintf(Out2File, "%g %.20g\n", 0.5 * (edge + outputEdges_h[iedge - 1]), sedge / ((double)(nedge)));
							}
							else{
								if(iedge > 0) fprintf(Out2File, "%g %.20g %.20g %.20g %d\n", 0.5 * (edge + outputEdges_h[iedge - 1]), sedge / ((double)(nedge)), param.T, P_h[iP], iedge - 1);
							}
							++iedge;
							nedge = 0;
							sedge = 0.0;
						}
					}
					++j;

					if(i >= ir - 1){
//printf("%d %d %d %d\n", ib, il, ir, Nxb);
						++ib;
						j = 0;
						if(param.doStoreK == 1){
							fprintf(Out2File,"\n\n");
						}
						if(param.doStoreK == 2 && ib < param.nbins){
							fclose(Out2File);
							sprintf(Out2Filename, "Out_%s_bin%.4d.dat", param.name, ib);
							Out2File = fopen(Out2Filename, "a");
						}
						iedge = 0;
					}
					if(ib >= param.nbins){
						break;
					}
				}
			}
		}//end of P loop
		fclose(Out2File);
	}
	//******************************
	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != 0){
		printf("Write error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}
	gettimeofday(&tt2, NULL);
	times = (tt2.tv_sec - tt1.tv_sec);
	timems = (tt2.tv_usec - tt1.tv_usec);

	time[7] = times + timems/1000000.0;
	printf("Time for write K(y):   %g seconds\n", time[7]);

	gettimeofday(&tt1, NULL);

	//set correction factor for simpsons rule needed for resampling
	SimpsonCoefficient();

	//*********************************
	//Calculate the Transmission function
	//*********************************
	if(param.doTransmission > 0 ){

		double *Tr_h, *Tr_d;
		Tr_h = (double*)malloc(param.nbins * param.nTr * sizeof(double));
		hipMalloc((void **) &Tr_d, param.nbins * param.nTr * sizeof(double));

		FILE *Out3File;
		char Out3Filename[160];

		if(param.doTransmission == 1){
			sprintf(Out3Filename, "Out_%s_tr.dat", param.name);
			Out3File = fopen(Out3Filename, filemode);
		}
		if(param.doTransmission == 2){
			if(param.replaceFiles == 1){
				for(int i = 0; i < param.nbins; ++i){
					sprintf(Out3Filename, "Out_%s_tr%.4d.dat", param.name, i);
					Out3File = fopen(Out3Filename, "w");
					fclose(Out3File);	
				}
			}
			sprintf(Out3Filename, "Out_%s_tr%.4d.dat", param.name, 0);
			Out3File = fopen(Out3Filename, "a");
		}

		for(int iP = 0; iP < param.nP; ++iP){
			if(param.doTransmission == 2 && iP > 0){
				fclose(Out3File);
				sprintf(Out3Filename, "Out_%s_tr%.4d.dat", param.name, 0);
				Out3File = fopen(Out3Filename, "a");
			}
			Integrate_kernel < 512 > <<< param.nbins, 512 >>> (K_d + iP * Nx, Tr_d, param.Nxb, param.nTr, param.dTr, Nxmin_d, param.kmin);
			hipMemcpy(Tr_h, Tr_d, param.nbins * param.nTr * sizeof(double), hipMemcpyDeviceToHost);
			for(int i = 0; i < param.nbins; ++i){
				for(int j = 0; j < param.nTr; ++j){
					double m = exp((j - param.nTr/2) * param.dTr);
					if(param.nP == 1){
						fprintf(Out3File, "%.20g %.20g\n", m, Tr_h[i * param.nTr + j]);
					}
					else{
						fprintf(Out3File, "%.20g %.20g %.20g %.20g %d\n", m, Tr_h[i * param.nTr + j], param.T, P_h[iP], j);
					}
				}
				if(param.doTransmission == 1){
					fprintf(Out3File, "\n\n");
				}
				if(param.doTransmission == 2 && i < param.nbins - 1){
					fclose(Out3File);
					sprintf(Out3Filename, "Out_%s_tr%.4d.dat", param.name, i + 1);
					Out3File = fopen(Out3Filename, "a");
				}
			}
		}
		fclose(Out3File);
		free(Tr_h);
		hipFree(Tr_d);
	}


	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != 0){
		printf("Transmission error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}
	gettimeofday(&tt2, NULL);
	times = (tt2.tv_sec - tt1.tv_sec);
	timems = (tt2.tv_usec - tt1.tv_usec);

	time[8] = times + timems/1000000.0;
	printf("Time for Transmission: %g seconds\n", time[8]);

	InfoFile = fopen(InfoFilename, "a");
	fprintf(InfoFile,"Time for write K(x):   %g seconds\n", time[3]);
	fprintf(InfoFile,"Time for mean K(x):    %g seconds\n", time[4]);
	fprintf(InfoFile,"Time for sort K(x):    %g seconds\n", time[5]);
	fprintf(InfoFile,"Time for Resampling:   %g seconds\n", time[6]);
	fprintf(InfoFile,"Time for write K(y):   %g seconds\n", time[7]);
	fprintf(InfoFile,"Time for Transmission: %g seconds\n", time[8]);
	fclose(InfoFile);	


	if(param.useHITEMP < 2) free_Line(L);
	else free2_Line(L);
	free(MaxLimits_h);
	free(K_h);
	free(x_h);
	free(Nxmin_h);
	free(outputEdges_h);
	free(binIndex_h);

	hipFree(K_d);
	hipFree(Nxmin_d);

	error = hipGetLastError();
	printf("Final error = %d = %s\n",error, hipGetErrorString(error));

	return 0;
}
